/*
 * BSD 2-Clause License
 *
 * Copyright (c) 2022, Christoph Neuhauser
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * * Redistributions of source code must retain the above copyright notice, this
 *   list of conditions and the following disclaimer.
 *
 * * Redistributions in binary form must reproduce the above copyright notice,
 *   this list of conditions and the following disclaimer in the documentation
 *   and/or other materials provided with the distribution.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <hip/hip_runtime.h>
#include <cstdint>

typedef unsigned uint32_t;

//#define USE_NORMALIZED_COORDINATES

extern "C" __global__ void memcpyFloatClampToZero(
        float* outputBuffer, const float* inputBuffer, uint32_t numElements) {
    uint32_t globalThreadIdx = blockIdx.x * blockDim.x + threadIdx.x;
    if (globalThreadIdx < numElements) {
        float value = inputBuffer[globalThreadIdx];
        if (value < 0.0f) {
            value = 0.0f;
        }
        outputBuffer[globalThreadIdx] = value;
    }
}

extern "C" __global__ void combineEnsembles(
        uint32_t xs, uint32_t ys, uint32_t zs, uint32_t es, uint32_t batchOffset, uint32_t batchSize,
        float minEnsembleVal, float maxEnsembleVal,
        float4* outputBuffer, hipTextureObject_t* scalarFieldEnsembles) {
    uint32_t globalThreadIdx = blockIdx.x * blockDim.x + threadIdx.x;
    if (globalThreadIdx >= batchSize) {
        return;
    }
    uint32_t pointIdxWriteOffset = globalThreadIdx * es;
    uint32_t pointIdxReadOffset = globalThreadIdx + batchOffset;
    uint32_t x = pointIdxReadOffset % xs;
    uint32_t y = (pointIdxReadOffset / xs) % ys;
    uint32_t z = pointIdxReadOffset / (xs * ys);
    float3 pointCoords = make_float3(
            2.0f * float(x) / float(xs - 1) - 1.0f,
            2.0f * float(y) / float(ys - 1) - 1.0f,
            2.0f * float(z) / float(zs - 1) - 1.0f);
    for (uint32_t e = 0; e < es; e++) {
        //float ensembleValue = tex3Dfetch(scalarFieldEnsembles[e], make_int4(x, y, z, 0)).x;
#ifdef USE_NORMALIZED_COORDINATES
        float ensembleValue = tex3D<float>(
                scalarFieldEnsembles[e],
                (float(x) + 0.5f) / float(xs),
                (float(y) + 0.5f) / float(ys),
                (float(z) + 0.5f) / float(zs));
#else
        float ensembleValue = tex3D<float>(scalarFieldEnsembles[e], float(x) + 0.5f, float(y) + 0.5f, float(z) + 0.5f);
#endif
        ensembleValue = (ensembleValue - minEnsembleVal) / (maxEnsembleVal - minEnsembleVal);
        outputBuffer[pointIdxWriteOffset + e] = make_float4(ensembleValue, pointCoords.x, pointCoords.y, pointCoords.z);
    }
}

extern "C" __global__ void combineEnsemblesReference(
        uint32_t xs, uint32_t ys, uint32_t zs, uint32_t es, uint3 referencePointIdx,
        float minEnsembleVal, float maxEnsembleVal,
        float4* outputBuffer, hipTextureObject_t* scalarFieldEnsembles) {
    uint32_t e = blockIdx.x * blockDim.x + threadIdx.x;
    if (e >= es) {
        return;
    }

    float3 pointCoords = make_float3(
            2.0f * float(referencePointIdx.x) / float(xs - 1) - 1.0f,
            2.0f * float(referencePointIdx.y) / float(ys - 1) - 1.0f,
            2.0f * float(referencePointIdx.z) / float(zs - 1) - 1.0f);
#ifdef USE_NORMALIZED_COORDINATES
    float ensembleValue = tex3D<float>(
                scalarFieldEnsembles[e],
                (float(referencePointIdx.x) + 0.5f) / float(xs),
                (float(referencePointIdx.y) + 0.5f) / float(ys),
                (float(referencePointIdx.z) + 0.5f) / float(zs));
#else
    float ensembleValue = tex3D<float>(
            scalarFieldEnsembles[e],
            float(referencePointIdx.x) + 0.5f,
            float(referencePointIdx.y) + 0.5f,
            float(referencePointIdx.z) + 0.5f);
#endif
    ensembleValue = (ensembleValue - minEnsembleVal) / (maxEnsembleVal - minEnsembleVal);
    outputBuffer[e] = make_float4(ensembleValue, pointCoords.x, pointCoords.y, pointCoords.z);
}

extern "C" __global__ void combineEnsemblesAligned(
        uint32_t xs, uint32_t ys, uint32_t zs, uint32_t es, uint32_t batchOffset, uint32_t batchSize,
        float minEnsembleVal, float maxEnsembleVal,
        float4* outputBuffer, hipTextureObject_t* scalarFieldEnsembles, uint32_t alignment) {
    uint32_t globalThreadIdx = blockIdx.x * blockDim.x + threadIdx.x;
    if (globalThreadIdx >= batchSize) {
        return;
    }
    uint32_t pointIdxWriteOffset = globalThreadIdx * es;
    uint32_t pointIdxReadOffset = globalThreadIdx + batchOffset;
    uint32_t x = pointIdxReadOffset % xs;
    uint32_t y = (pointIdxReadOffset / xs) % ys;
    uint32_t z = pointIdxReadOffset / (xs * ys);
    float3 pointCoords = make_float3(
            2.0f * float(x) / float(xs - 1) - 1.0f,
            2.0f * float(y) / float(ys - 1) - 1.0f,
            2.0f * float(z) / float(zs - 1) - 1.0f);
    for (uint32_t e = 0; e < es; e++) {
        //float ensembleValue = tex3Dfetch(scalarFieldEnsembles[e], make_int4(x, y, z, 0)).x;
#ifdef USE_NORMALIZED_COORDINATES
        float ensembleValue = tex3D<float>(
                scalarFieldEnsembles[e],
                (float(x) + 0.5f) / float(xs),
                (float(y) + 0.5f) / float(ys),
                (float(z) + 0.5f) / float(zs));
#else
        float ensembleValue = tex3D<float>(scalarFieldEnsembles[e], float(x) + 0.5f, float(y) + 0.5f, float(z) + 0.5f);
#endif
        ensembleValue = (ensembleValue - minEnsembleVal) / (maxEnsembleVal - minEnsembleVal);
        outputBuffer[(pointIdxWriteOffset + e) * alignment] =
                make_float4(ensembleValue, pointCoords.x, pointCoords.y, pointCoords.z);
    }
}

extern "C" __global__ void combineEnsemblesReferenceAligned(
        uint32_t xs, uint32_t ys, uint32_t zs, uint32_t es, uint3 referencePointIdx,
        float minEnsembleVal, float maxEnsembleVal,
        float4* outputBuffer, hipTextureObject_t* scalarFieldEnsembles, uint32_t alignment) {
    uint32_t e = blockIdx.x * blockDim.x + threadIdx.x;
    if (e >= es) {
        return;
    }

    float3 pointCoords = make_float3(
            2.0f * float(referencePointIdx.x) / float(xs - 1) - 1.0f,
            2.0f * float(referencePointIdx.y) / float(ys - 1) - 1.0f,
            2.0f * float(referencePointIdx.z) / float(zs - 1) - 1.0f);
#ifdef USE_NORMALIZED_COORDINATES
    float ensembleValue = tex3D<float>(
                scalarFieldEnsembles[e],
                (float(referencePointIdx.x) + 0.5f) / float(xs),
                (float(referencePointIdx.y) + 0.5f) / float(ys),
                (float(referencePointIdx.z) + 0.5f) / float(zs));
#else
    float ensembleValue = tex3D<float>(
            scalarFieldEnsembles[e],
            float(referencePointIdx.x) + 0.5f,
            float(referencePointIdx.y) + 0.5f,
            float(referencePointIdx.z) + 0.5f);
#endif
    ensembleValue = (ensembleValue - minEnsembleVal) / (maxEnsembleVal - minEnsembleVal);
    outputBuffer[e * alignment] = make_float4(ensembleValue, pointCoords.x, pointCoords.y, pointCoords.z);
}

extern "C" __global__ void writeGridPositions(
        uint32_t xs, uint32_t ys, uint32_t zs, uint32_t batchOffset, uint32_t batchSize, float* outputBuffer,
        uint32_t stride) {
    uint32_t globalThreadIdx = blockIdx.x * blockDim.x + threadIdx.x;
    if (globalThreadIdx >= batchSize) {
        return;
    }
    uint32_t pointIdxWriteOffset = globalThreadIdx * stride;
    uint32_t pointIdxReadOffset = globalThreadIdx + batchOffset;
    uint32_t x = pointIdxReadOffset % xs;
    uint32_t y = (pointIdxReadOffset / xs) % ys;
    uint32_t z = pointIdxReadOffset / (xs * ys);
    outputBuffer[pointIdxWriteOffset] = 2.0f * float(x) / float(xs - 1) - 1.0f;
    outputBuffer[pointIdxWriteOffset + 1] = 2.0f * float(y) / float(ys - 1) - 1.0f;
    outputBuffer[pointIdxWriteOffset + 2] = 2.0f * float(z) / float(zs - 1) - 1.0f;
}

extern "C" __global__ void writeGridPositionReference(
        uint32_t xs, uint32_t ys, uint32_t zs, uint3 referencePointIdx, float* outputBuffer) {
    uint32_t globalThreadIdx = blockIdx.x * blockDim.x + threadIdx.x;
    if (globalThreadIdx >= 1) {
        return;
    }
    outputBuffer[0] = 2.0f * float(referencePointIdx.x) / float(xs - 1) - 1.0f;
    outputBuffer[1] = 2.0f * float(referencePointIdx.y) / float(ys - 1) - 1.0f;
    outputBuffer[2] = 2.0f * float(referencePointIdx.z) / float(zs - 1) - 1.0f;
}
