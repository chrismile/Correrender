#include "hip/hip_runtime.h"
/*
 * BSD 2-Clause License
 *
 * Copyright (c) 2021, Christoph Neuhauser
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * * Redistributions of source code must retain the above copyright notice, this
 *   list of conditions and the following disclaimer.
 *
 * * Redistributions in binary form must reproduce the above copyright notice,
 *   this list of conditions and the following disclaimer in the documentation
 *   and/or other materials provided with the distribution.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <hipblas.h>
#include <hipsolver.h>

#include "CudaHelpers.cuh"
#include "CudaSolver.hpp"

static bool useCustomCudaStream = false;
static hipStream_t stream = nullptr;
static hipsolverHandle_t cusolverHandle = nullptr;
static hipblasHandle_t cublasHandle = nullptr;

void cudaInit(void* cudaStream) {
    // Initialize cuBLAS and cuSOLVER.
    if (cudaStream) {
        stream = hipStream_t(cudaStream);
        useCustomCudaStream = true;
    } else {
        cudaErrorCheck(hipStreamCreate(&stream));
    }
    cudaErrorCheck(hipsolverDnCreate(&cusolverHandle));
    cudaErrorCheck(hipblasCreate(&cublasHandle));
    cudaErrorCheck(hipsolverSetStream(cusolverHandle, stream));
    cudaErrorCheck(hipblasSetStream(cublasHandle, stream));
}

void cudaRelease() {
    // Free cuBLAS and cuSOLVER.
    if (cusolverHandle) {
        cudaErrorCheck(hipsolverDnDestroy(cusolverHandle));
        cusolverHandle = nullptr;
    }
    if (cublasHandle) {
        cudaErrorCheck(hipblasDestroy(cublasHandle));
        cublasHandle = nullptr;
    }
    if (stream) {
        if (!useCustomCudaStream) {
            cudaErrorCheck(hipStreamDestroy(stream));
        }
        stream = nullptr;
    }
}

// https://eigen.tuxfamily.org/dox/TopicCUDA.html
// https://docs.nvidia.com/cuda/cusolver/index.html#introduction
void solveSystemOfLinearEquationsCuda(
        CudaSolverType cudaSolverType, bool useRelaxation, const Real lambdaL,
        const Eigen::MatrixXr& A, const Eigen::MatrixXr& b, Eigen::MatrixXr& x) {
    // A \in R^NxL, I \in R^NxD1, l \in R^LxD1, A*l = I.
    const int N = int(A.rows());
    constexpr int L = 9; //A.cols();
    constexpr int D1 = 1; //I.cols();
    assert(A.rows() == b.rows());
    assert(L == A.cols());//assert(L == 9);
    assert(D1 == b.cols());//assert(D1 == 1);
    x = Eigen::Matrix<Real, L, D1>();

    Eigen::MatrixXr M_I = Eigen::Matrix<Real, L, L>::Identity();

    // Allocate memory on the device.
    Real* dA = nullptr;
    cudaErrorCheck(hipMalloc((void**)&dA, sizeof(Real) * N * L));
    cudaErrorCheck(hipMemcpy(dA, A.data(), sizeof(Real) * N * L, hipMemcpyHostToDevice));
    Real* db = nullptr;
    cudaErrorCheck(hipMalloc((void**)&db, sizeof(Real) * N * D1));
    cudaErrorCheck(hipMemcpy(db, b.data(), sizeof(Real) * N * D1, hipMemcpyHostToDevice));
    Real* dx = nullptr;
    cudaErrorCheck(hipMalloc((void**)&dx, sizeof(Real) * L * D1));

    // lhs = A^T*A + lambda_l*M_I
    Real* dLhs = nullptr;
    cudaErrorCheck(hipMalloc((void**)&dLhs, sizeof(Real) * L * L));
    hipMemcpy(dLhs, M_I.data(), sizeof(Real) * L * L, hipMemcpyHostToDevice);
    // rhs = A^T*b
    Real* dRhs = nullptr;
    cudaErrorCheck(hipMalloc((void**)&dRhs, sizeof(Real) * L * D1));

    const int lda = N;
    const int ldb = N;
    constexpr int ldLhs = L;
    constexpr int ldRhs = L;

    // https://docs.nvidia.com/cuda/cublas/index.html#cublas-lt-t-gt-gemm
    // hipblasSgemm: C = alpha*op(A)op(B) + beta*C
    const Real alpha = Real(1.0);
    // Compute: lhs = A^T*A + lambda_l*M_I.
    const Real beta0 = lambdaL;
    cudaErrorCheck(cublasRgemm(
            cublasHandle, HIPBLAS_OP_T, HIPBLAS_OP_N,
            L, L, N, &alpha, dA, lda, dA, lda, &beta0, dLhs, ldLhs));
    // Compute: rhs = A^T*b.
    const Real beta1 = Real(0.0);
    cudaErrorCheck(cublasRgemm(
            cublasHandle, HIPBLAS_OP_T, HIPBLAS_OP_N,
            L, D1, N, &alpha, dA, lda, db, ldb, &beta1, dRhs, ldRhs));

    int lwork = 0;
    Real* dWork = nullptr;
    int* dInfo = nullptr;
    int hInfo = 0;
    cudaErrorCheck(hipMalloc ((void**)&dInfo, sizeof(int)));

    // Now, solve lhs*l = rhs.
    switch(cudaSolverType) {
        // See: https://docs.nvidia.com/cuda/cusolver/index.html#lu_examples
        case CudaSolverType::LU: case CudaSolverType::LU_PIVOT: {
            bool usePivot = cudaSolverType == CudaSolverType::LU_PIVOT;

            // Query working space required by getrf.
            cudaErrorCheck(cusolverDnRgetrf_bufferSize(
                    cusolverHandle, L, L, dLhs, ldLhs, &lwork));
            cudaErrorCheck(hipMalloc((void**)&dWork, sizeof(Real)*lwork));

            // Create data for pivot if user requests pivoting.
            int* dbpiv = nullptr;
            if (usePivot) {
                cudaErrorCheck(hipMalloc((void**)&dbpiv, sizeof(int) * L));
            }

            // LU factorization.
            cudaErrorCheck(cusolverDnRgetrf(
                    cusolverHandle, L, L, dLhs, ldLhs, dWork, dbpiv, dInfo));
            cudaErrorCheck(hipDeviceSynchronize());

            // Check whether the factorization was successful.
            cudaErrorCheck(hipMemcpy(&hInfo, dInfo, sizeof(int), hipMemcpyDeviceToHost));
            if (hInfo != 0) {
                if (usePivot) {
                    std::cerr << "ERROR: CudaSolverType::LU_PIVOT failed!" << std::endl;
                } else {
                    std::cerr << "ERROR: CudaSolverType::LU failed!" << std::endl;
                }
                exit(1);
            }

            // Solve A*l = LU*l = I.
            cudaErrorCheck(cusolverDnRgetrs(
                    cusolverHandle, HIPBLAS_OP_N, L, D1, dLhs, ldLhs, dbpiv, dRhs, ldRhs, dInfo));
            cudaErrorCheck(hipDeviceSynchronize());
            cudaErrorCheck(hipMemcpy(x.data(), dRhs, sizeof(Real) * L * D1, hipMemcpyDeviceToHost));

            if (usePivot && dbpiv) {
                cudaErrorCheck(hipFree(dbpiv));
            }

            break;
        }

        // See: https://docs.nvidia.com/cuda/cusolver/index.html#qr_examples
        case CudaSolverType::QR: {
            const Real one = 1.0f;
            int lwork_geqrf = 0;
            int lwork_ormqr = 0;
            Real* dTau = nullptr;

            cudaErrorCheck(hipMalloc((void**)&dTau, sizeof(Real) * L));

            // Query working space required by geqrf and ormqr.
            cudaErrorCheck(cusolverDnRgeqrf_bufferSize(
                    cusolverHandle, L, L, dLhs, ldLhs, &lwork_geqrf));
            cudaErrorCheck(cusolverDnRormqr_bufferSize(
                    cusolverHandle, HIPBLAS_SIDE_LEFT, HIPBLAS_OP_T,
                    L, D1, L, dLhs, ldLhs, dTau, dRhs, ldRhs, &lwork_ormqr));
            lwork = std::max(lwork_geqrf, lwork_ormqr);
            cudaErrorCheck(hipMalloc((void**)&dWork, sizeof(Real)*lwork));

            // Compute the QR factorization.
            cudaErrorCheck(cusolverDnRgeqrf(
                    cusolverHandle, L, L, dLhs, ldLhs, dTau, dWork, lwork, dInfo));
            cudaErrorCheck(hipDeviceSynchronize());

            // Check whether the factorization was successful.
            cudaErrorCheck(hipMemcpy(&hInfo, dInfo, sizeof(int), hipMemcpyDeviceToHost));
            if (hInfo != 0) {
                std::cerr << "ERROR: CudaSolverType::QR failed!" << std::endl;
                exit(1);
            }

            // Compute Q^T*I.
            cudaErrorCheck(cusolverDnRormqr(
                    cusolverHandle, HIPBLAS_SIDE_LEFT, HIPBLAS_OP_T,
                    L, D1, L, dLhs, ldLhs, dTau, dRhs, ldRhs, dWork, lwork, dInfo));
            cudaErrorCheck(hipDeviceSynchronize());

            // Check whether the factorization was successful.
            cudaErrorCheck(hipMemcpy(&hInfo, dInfo, sizeof(int), hipMemcpyDeviceToHost));
            if (hInfo != 0) {
                std::cerr << "ERROR: CudaSolverType::QR failed!" << std::endl;
                exit(1);
            }

            // Solve R*l = Q^T*I (i.e., l = R \ Q^T*I).
            cudaErrorCheck(cublasRtrsm(
                    cublasHandle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT,
                    L, D1, &one, dLhs, ldLhs, dRhs, ldRhs));
            cudaErrorCheck(hipDeviceSynchronize());
            cudaErrorCheck(hipMemcpy(x.data(), dRhs, sizeof(Real) * L * D1, hipMemcpyDeviceToHost));

            if (dTau) {
                cudaErrorCheck(hipFree(dTau));
            }

            break;
        }

        // https://docs.nvidia.com/cuda/cusolver/index.html#chol_examples
        case CudaSolverType::CHOL: {
            const hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;

            // Query working space required by potrf.
            cudaErrorCheck(cusolverDnRpotrf_bufferSize(
                    cusolverHandle, uplo, L, dLhs, ldLhs, &lwork));
            cudaErrorCheck(hipMalloc((void**)&dWork, sizeof(Real)*lwork));

            // Cholesky factorization.
            cudaErrorCheck(cusolverDnRpotrf(
                    cusolverHandle, uplo, L, dLhs, ldLhs, dWork, lwork, dInfo));
            cudaErrorCheck(hipDeviceSynchronize());

            // Check whether the factorization was successful.
            cudaErrorCheck(hipMemcpy(&hInfo, dInfo, sizeof(int), hipMemcpyDeviceToHost));
            if (hInfo != 0) {
                std::cerr << "ERROR: CudaSolverType::CHOL failed!" << std::endl;
                exit(1);
            }

            // Solving step.
            cudaErrorCheck(cusolverDnRpotrs(
                    cusolverHandle, uplo, L, D1, dLhs, ldLhs, dRhs, ldRhs, dInfo));
            cudaErrorCheck(hipDeviceSynchronize());

            // Check whether the solving step was successful.
            cudaErrorCheck(hipMemcpy(&hInfo, dInfo, sizeof(int), hipMemcpyDeviceToHost));
            if (hInfo != 0) {
                std::cerr << "ERROR: CudaSolverType::CHOL failed!" << std::endl;
                exit(1);
            }

            cudaErrorCheck(hipMemcpy(x.data(), dRhs, sizeof(Real) * L * D1, hipMemcpyDeviceToHost));

            break;
        }
    }

    // Free the allocated memory.
    if (dWork) {
        cudaErrorCheck(hipFree(dWork));
    }
    if (dInfo) {
        cudaErrorCheck(hipFree(dInfo));
    }
    if (dLhs) {
        cudaErrorCheck(hipFree(dLhs));
    }
    if (dRhs) {
        cudaErrorCheck(hipFree(dRhs));
    }
    if (dA) {
        cudaErrorCheck(hipFree(dA));
    }
    if (db) {
        cudaErrorCheck(hipFree(db));
    }
    if (dx) {
        cudaErrorCheck(hipFree(dx));
    }
}
