#include "hip/hip_runtime.h"
/*
 * BSD 2-Clause License
 *
 * Copyright (c) 2021, Christoph Neuhauser
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * * Redistributions of source code must retain the above copyright notice, this
 *   list of conditions and the following disclaimer.
 *
 * * Redistributions in binary form must reproduce the above copyright notice,
 *   this list of conditions and the following disclaimer in the documentation
 *   and/or other materials provided with the distribution.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <hipblas.h>
#include <hipsolver.h>

#include <Graphics/Vulkan/Utils/InteropCuda.hpp>

#include "CudaHelpers.cuh"
#include "CudaSolver.hpp"

static bool useCustomCudaStream = false;
static hipCtx_t cuContext = nullptr;
static hipStream_t cuStream = nullptr;
static hipStream_t stream = nullptr;
static hipsolverHandle_t cusolverHandle = nullptr;
static hipblasHandle_t cublasHandle = nullptr;

void cudaInit(void* cudaStream) {
    // Initialize cuBLAS and cuSOLVER.
    if (cudaStream) {
        stream = hipStream_t(cudaStream);
        useCustomCudaStream = true;
    } else {
        //cudaErrorCheck(hipStreamCreate(&stream));
        sgl::vk::checkCUresult(sgl::vk::g_cudaDeviceApiFunctionTable.hipCtxCreate(
                &cuContext, hipDeviceScheduleAuto, 0), "Error in hipCtxCreate: ");
        sgl::vk::checkCUresult(sgl::vk::g_cudaDeviceApiFunctionTable.hipStreamCreateWithFlags(
                &cuStream, 0), "Error in hipStreamCreateWithFlags: ");
        stream = cuStream;
    }
    cudaErrorCheck(hipsolverDnCreate(&cusolverHandle));
    cudaErrorCheck(hipblasCreate(&cublasHandle));
    cudaErrorCheck(hipsolverSetStream(cusolverHandle, stream));
    cudaErrorCheck(hipblasSetStream(cublasHandle, stream));
}

void cudaRelease() {
    // Free cuBLAS and cuSOLVER.
    if (cusolverHandle) {
        cudaErrorCheck(hipsolverDnDestroy(cusolverHandle));
        cusolverHandle = nullptr;
    }
    if (cublasHandle) {
        cudaErrorCheck(hipblasDestroy(cublasHandle));
        cublasHandle = nullptr;
    }
    if (stream) {
        if (!useCustomCudaStream) {
            //cudaErrorCheck(hipStreamDestroy(stream));
            sgl::vk::checkCUresult(sgl::vk::g_cudaDeviceApiFunctionTable.hipStreamDestroy(
                    cuStream), "Error in hipStreamDestroy: ");
            sgl::vk::checkCUresult(sgl::vk::g_cudaDeviceApiFunctionTable.hipCtxDestroy(
                    cuContext), "Error in hipCtxDestroy: ");
        }
        stream = nullptr;
        cuStream = nullptr;
        cuContext = nullptr;
    }
}

// https://eigen.tuxfamily.org/dox/TopicCUDA.html
// https://docs.nvidia.com/cuda/cusolver/index.html#introduction
void solveSystemOfLinearEquationsCuda(
        CudaSolverType cudaSolverType, bool useRelaxation, const Real lambdaL,
        const Eigen::MatrixXr& A, const Eigen::MatrixXr& b, Eigen::MatrixXr& x) {
    // A \in R^NxL, I \in R^Nx1, l \in R^Lx1, A*l = I.
    const int N = int(A.rows());
    const int L = int(A.cols());
    assert(A.rows() == b.rows());
    assert(1 == b.cols());
    x = Eigen::MatrixXr(L, 1);

    Eigen::MatrixXr M_I = Eigen::MatrixXr::Identity(L, L);

    // Allocate memory on the device.
    Real* dA = nullptr;
    cudaErrorCheck(hipMalloc((void**)&dA, sizeof(Real) * N * L));
    cudaErrorCheck(hipMemcpy(dA, A.data(), sizeof(Real) * N * L, hipMemcpyHostToDevice));
    Real* db = nullptr;
    cudaErrorCheck(hipMalloc((void**)&db, sizeof(Real) * N));
    cudaErrorCheck(hipMemcpy(db, b.data(), sizeof(Real) * N, hipMemcpyHostToDevice));
    Real* dx = nullptr;
    cudaErrorCheck(hipMalloc((void**)&dx, sizeof(Real) * L));

    // lhs = A^T*A + lambda_l*M_I
    Real* dLhs = nullptr;
    cudaErrorCheck(hipMalloc((void**)&dLhs, sizeof(Real) * L * L));
    hipMemcpy(dLhs, M_I.data(), sizeof(Real) * L * L, hipMemcpyHostToDevice);
    // rhs = A^T*b
    Real* dRhs = nullptr;
    cudaErrorCheck(hipMalloc((void**)&dRhs, sizeof(Real) * L));

    const int lda = N;
    const int ldb = N;
    const int ldLhs = L;
    const int ldRhs = L;

    // https://docs.nvidia.com/cuda/cublas/index.html#cublas-lt-t-gt-gemm
    // hipblasSgemm: C = alpha*op(A)op(B) + beta*C
    const Real alpha = Real(1.0);
    // Compute: lhs = A^T*A + lambda_l*M_I.
    const Real beta0 = lambdaL;
    cudaErrorCheck(cublasRgemm(
            cublasHandle, HIPBLAS_OP_T, HIPBLAS_OP_N,
            L, L, N, &alpha, dA, lda, dA, lda, &beta0, dLhs, ldLhs));
    // Compute: rhs = A^T*b.
    const Real beta1 = Real(0.0);
    cudaErrorCheck(cublasRgemm(
            cublasHandle, HIPBLAS_OP_T, HIPBLAS_OP_N,
            L, 1, N, &alpha, dA, lda, db, ldb, &beta1, dRhs, ldRhs));

    int lwork = 0;
    Real* dWork = nullptr;
    int* dInfo = nullptr;
    int hInfo = 0;
    cudaErrorCheck(hipMalloc ((void**)&dInfo, sizeof(int)));

    // Now, solve lhs*l = rhs.
    switch(cudaSolverType) {
        // See: https://docs.nvidia.com/cuda/cusolver/index.html#lu_examples
        case CudaSolverType::LU: case CudaSolverType::LU_PIVOT: {
            bool usePivot = cudaSolverType == CudaSolverType::LU_PIVOT;

            // Query working space required by getrf.
            cudaErrorCheck(cusolverDnRgetrf_bufferSize(
                    cusolverHandle, L, L, dLhs, ldLhs, &lwork));
            cudaErrorCheck(hipMalloc((void**)&dWork, sizeof(Real)*lwork));

            // Create data for pivot if user requests pivoting.
            int* dbpiv = nullptr;
            if (usePivot) {
                cudaErrorCheck(hipMalloc((void**)&dbpiv, sizeof(int) * L));
            }

            // LU factorization.
            cudaErrorCheck(cusolverDnRgetrf(
                    cusolverHandle, L, L, dLhs, ldLhs, dWork, dbpiv, dInfo));
            cudaErrorCheck(hipDeviceSynchronize());

            // Check whether the factorization was successful.
            cudaErrorCheck(hipMemcpy(&hInfo, dInfo, sizeof(int), hipMemcpyDeviceToHost));
            if (hInfo != 0) {
                if (usePivot) {
                    std::cerr << "ERROR: CudaSolverType::LU_PIVOT failed!" << std::endl;
                } else {
                    std::cerr << "ERROR: CudaSolverType::LU failed!" << std::endl;
                }
                exit(1);
            }

            // Solve A*l = LU*l = I.
            cudaErrorCheck(cusolverDnRgetrs(
                    cusolverHandle, HIPBLAS_OP_N, L, 1, dLhs, ldLhs, dbpiv, dRhs, ldRhs, dInfo));
            cudaErrorCheck(hipDeviceSynchronize());
            cudaErrorCheck(hipMemcpy(x.data(), dRhs, sizeof(Real) * L, hipMemcpyDeviceToHost));

            if (usePivot && dbpiv) {
                cudaErrorCheck(hipFree(dbpiv));
            }

            break;
        }

        // See: https://docs.nvidia.com/cuda/cusolver/index.html#qr_examples
        case CudaSolverType::QR: {
            const Real one = 1.0f;
            int lwork_geqrf = 0;
            int lwork_ormqr = 0;
            Real* dTau = nullptr;

            cudaErrorCheck(hipMalloc((void**)&dTau, sizeof(Real) * L));

            // Query working space required by geqrf and ormqr.
            cudaErrorCheck(cusolverDnRgeqrf_bufferSize(
                    cusolverHandle, L, L, dLhs, ldLhs, &lwork_geqrf));
            cudaErrorCheck(cusolverDnRormqr_bufferSize(
                    cusolverHandle, HIPBLAS_SIDE_LEFT, HIPBLAS_OP_T,
                    L, 1, L, dLhs, ldLhs, dTau, dRhs, ldRhs, &lwork_ormqr));
            lwork = std::max(lwork_geqrf, lwork_ormqr);
            cudaErrorCheck(hipMalloc((void**)&dWork, sizeof(Real)*lwork));

            // Compute the QR factorization.
            cudaErrorCheck(cusolverDnRgeqrf(
                    cusolverHandle, L, L, dLhs, ldLhs, dTau, dWork, lwork, dInfo));
            cudaErrorCheck(hipDeviceSynchronize());

            // Check whether the factorization was successful.
            cudaErrorCheck(hipMemcpy(&hInfo, dInfo, sizeof(int), hipMemcpyDeviceToHost));
            if (hInfo != 0) {
                std::cerr << "ERROR: CudaSolverType::QR failed!" << std::endl;
                exit(1);
            }

            // Compute Q^T*I.
            cudaErrorCheck(cusolverDnRormqr(
                    cusolverHandle, HIPBLAS_SIDE_LEFT, HIPBLAS_OP_T,
                    L, 1, L, dLhs, ldLhs, dTau, dRhs, ldRhs, dWork, lwork, dInfo));
            cudaErrorCheck(hipDeviceSynchronize());

            // Check whether the factorization was successful.
            cudaErrorCheck(hipMemcpy(&hInfo, dInfo, sizeof(int), hipMemcpyDeviceToHost));
            if (hInfo != 0) {
                std::cerr << "ERROR: CudaSolverType::QR failed!" << std::endl;
                exit(1);
            }

            // Solve R*l = Q^T*I (i.e., l = R \ Q^T*I).
            cudaErrorCheck(cublasRtrsm(
                    cublasHandle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT,
                    L, 1, &one, dLhs, ldLhs, dRhs, ldRhs));
            cudaErrorCheck(hipDeviceSynchronize());
            cudaErrorCheck(hipMemcpy(x.data(), dRhs, sizeof(Real) * L, hipMemcpyDeviceToHost));

            if (dTau) {
                cudaErrorCheck(hipFree(dTau));
            }

            break;
        }

        // https://docs.nvidia.com/cuda/cusolver/index.html#chol_examples
        case CudaSolverType::CHOL: {
            const hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;

            // Query working space required by potrf.
            cudaErrorCheck(cusolverDnRpotrf_bufferSize(
                    cusolverHandle, uplo, L, dLhs, ldLhs, &lwork));
            cudaErrorCheck(hipMalloc((void**)&dWork, sizeof(Real)*lwork));

            // Cholesky factorization.
            cudaErrorCheck(cusolverDnRpotrf(
                    cusolverHandle, uplo, L, dLhs, ldLhs, dWork, lwork, dInfo));
            cudaErrorCheck(hipDeviceSynchronize());

            // Check whether the factorization was successful.
            cudaErrorCheck(hipMemcpy(&hInfo, dInfo, sizeof(int), hipMemcpyDeviceToHost));
            if (hInfo != 0) {
                std::cerr << "ERROR: CudaSolverType::CHOL failed!" << std::endl;
                exit(1);
            }

            // Solving step.
            cudaErrorCheck(cusolverDnRpotrs(
                    cusolverHandle, uplo, L, 1, dLhs, ldLhs, dRhs, ldRhs, dInfo));
            cudaErrorCheck(hipDeviceSynchronize());

            // Check whether the solving step was successful.
            cudaErrorCheck(hipMemcpy(&hInfo, dInfo, sizeof(int), hipMemcpyDeviceToHost));
            if (hInfo != 0) {
                std::cerr << "ERROR: CudaSolverType::CHOL failed!" << std::endl;
                exit(1);
            }

            cudaErrorCheck(hipMemcpy(x.data(), dRhs, sizeof(Real) * L, hipMemcpyDeviceToHost));

            break;
        }
    }

    // Free the allocated memory.
    if (dWork) {
        cudaErrorCheck(hipFree(dWork));
    }
    if (dInfo) {
        cudaErrorCheck(hipFree(dInfo));
    }
    if (dLhs) {
        cudaErrorCheck(hipFree(dLhs));
    }
    if (dRhs) {
        cudaErrorCheck(hipFree(dRhs));
    }
    if (dA) {
        cudaErrorCheck(hipFree(dA));
    }
    if (db) {
        cudaErrorCheck(hipFree(db));
    }
    if (dx) {
        cudaErrorCheck(hipFree(dx));
    }
}
