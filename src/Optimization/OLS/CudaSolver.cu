#include "hip/hip_runtime.h"
/*
 * BSD 2-Clause License
 *
 * Copyright (c) 2021, Christoph Neuhauser
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * * Redistributions of source code must retain the above copyright notice, this
 *   list of conditions and the following disclaimer.
 *
 * * Redistributions in binary form must reproduce the above copyright notice,
 *   this list of conditions and the following disclaimer in the documentation
 *   and/or other materials provided with the distribution.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <hipblas.h>
#include <hipsolver.h>
#include <hipsolver.h>

#include <Graphics/Vulkan/Utils/InteropCuda.hpp>

#include "CudaHelpers.cuh"
//#include "cgls.cuh"
#include "CudaSolver.hpp"

static bool useCustomCudaStream = false;
static hipCtx_t cuContext = nullptr;
static hipStream_t cuStream = nullptr;
static hipStream_t stream = nullptr;
static hipsolverHandle_t cusolverHandle = nullptr;
static hipsolverSpHandle_t cusolverSpHandle = nullptr;
static hipblasHandle_t cublasHandle = nullptr;

void cudaInit(void* cudaStream) {
    // Initialize cuBLAS and cuSOLVER.
    if (cudaStream) {
        stream = hipStream_t(cudaStream);
        useCustomCudaStream = true;
    } else {
        //cudaErrorCheck(hipStreamCreate(&stream));
        sgl::vk::checkCUresult(sgl::vk::g_cudaDeviceApiFunctionTable.hipCtxCreate(
                &cuContext, hipDeviceScheduleAuto, 0), "Error in hipCtxCreate: ");
        sgl::vk::checkCUresult(sgl::vk::g_cudaDeviceApiFunctionTable.hipStreamCreateWithFlags(
                &cuStream, 0), "Error in hipStreamCreateWithFlags: ");
        stream = cuStream;
    }
    cudaErrorCheck(hipsolverDnCreate(&cusolverHandle));
    cudaErrorCheck(hipsolverSpCreate(&cusolverSpHandle));
    cudaErrorCheck(hipblasCreate(&cublasHandle));
    cudaErrorCheck(hipsolverSetStream(cusolverHandle, stream));
    cudaErrorCheck(hipsolverSpSetStream(cusolverSpHandle, stream));
    cudaErrorCheck(hipblasSetStream(cublasHandle, stream));
}

void cudaRelease() {
    // Free cuBLAS and cuSOLVER.
    if (cusolverHandle) {
        cudaErrorCheck(hipsolverDnDestroy(cusolverHandle));
        cusolverHandle = nullptr;
    }
    if (cusolverSpHandle) {
        cudaErrorCheck(hipsolverSpDestroy(cusolverSpHandle));
        cusolverSpHandle = nullptr;
    }
    if (cublasHandle) {
        cudaErrorCheck(hipblasDestroy(cublasHandle));
        cublasHandle = nullptr;
    }
    if (stream) {
        if (!useCustomCudaStream) {
            //cudaErrorCheck(hipStreamDestroy(stream));
            sgl::vk::checkCUresult(sgl::vk::g_cudaDeviceApiFunctionTable.hipStreamDestroy(
                    cuStream), "Error in hipStreamDestroy: ");
            sgl::vk::checkCUresult(sgl::vk::g_cudaDeviceApiFunctionTable.hipCtxDestroy(
                    cuContext), "Error in hipCtxDestroy: ");
        }
        stream = nullptr;
        cuStream = nullptr;
        cuContext = nullptr;
    }
}

// https://eigen.tuxfamily.org/dox/TopicCUDA.html
// https://docs.nvidia.com/cuda/cusolver/index.html#introduction
void solveLeastSquaresCudaDense(
        CudaSolverType cudaSolverType, bool useNormalEquations, const Real lambdaL,
        const Eigen::MatrixXr& A, const Eigen::MatrixXr& b, Eigen::MatrixXr& x) {
    if (!useNormalEquations && cudaSolverType != CudaSolverType::QR) {
        sgl::Logfile::get()->writeError(
                "Error in solveLeastSquaresCudaDense: Only QR can solve non-square matrices. Switching to QR.");
        cudaSolverType = CudaSolverType::QR;
    }

    // A \in R^MxN, I \in R^Nx1, l \in R^Lx1, A*l = I.
    const int M = int(A.rows());
    const int N = int(A.cols());
    assert(A.rows() == b.rows());
    assert(1 == b.cols());
    x = Eigen::MatrixXr(N, 1);

    // Allocate memory on the device.
    Real* dA = nullptr;
    cudaErrorCheck(hipMalloc((void**)&dA, sizeof(Real) * M * N));
    cudaErrorCheck(hipMemcpy(dA, A.data(), sizeof(Real) * M * N, hipMemcpyHostToDevice));
    Real* db = nullptr;
    cudaErrorCheck(hipMalloc((void**)&db, sizeof(Real) * M));
    cudaErrorCheck(hipMemcpy(db, b.data(), sizeof(Real) * M, hipMemcpyHostToDevice));
    Real* dx = nullptr;
    cudaErrorCheck(hipMalloc((void**)&dx, sizeof(Real) * N));

    // lhs = A^T*A + lambda_l*M_I
    Real* dLhs = nullptr;
    // rhs = A^T*b
    Real* dRhs = nullptr;
    if (useNormalEquations) {
        Eigen::MatrixXr M_I = Eigen::MatrixXr::Identity(N, N);
        cudaErrorCheck(hipMalloc((void**)&dLhs, sizeof(Real) * N * N));
        hipMemcpy(dLhs, M_I.data(), sizeof(Real) * N * N, hipMemcpyHostToDevice);
        cudaErrorCheck(hipMalloc((void**)&dRhs, sizeof(Real) * N));
    } else {
        dLhs = dA;
        dRhs = db;
    }

    const int lda = M;
    const int ldb = M;
    const int lhsM = useNormalEquations ? N : M;
    const int lhsN = N;
    const int ldLhs = useNormalEquations ? N : M;
    const int ldRhs = useNormalEquations ? N : M;

    if (useNormalEquations) {
        // https://docs.nvidia.com/cuda/cublas/index.html#cublas-lt-t-gt-gemm
        // hipblasSgemm: C = alpha*op(A)op(B) + beta*C
        const Real alpha = Real(1.0);
        // Compute: lhs = A^T*A + lambda_l*M_I.
        const Real beta0 = lambdaL;
        cudaErrorCheck(cublasRgemm(
                cublasHandle, HIPBLAS_OP_T, HIPBLAS_OP_N,
                N, N, M, &alpha, dA, lda, dA, lda, &beta0, dLhs, ldLhs));
        // Compute: rhs = A^T*b.
        const Real beta1 = Real(0.0);
        cudaErrorCheck(cublasRgemm(
                cublasHandle, HIPBLAS_OP_T, HIPBLAS_OP_N,
                N, 1, M, &alpha, dA, lda, db, ldb, &beta1, dRhs, ldRhs));
    }

    int lwork = 0;
    Real* dWork = nullptr;
    int* dInfo = nullptr;
    int hInfo = 0;
    cudaErrorCheck(hipMalloc((void**)&dInfo, sizeof(int)));

    // Now, solve lhs*x = rhs.
    switch(cudaSolverType) {
        // See: https://docs.nvidia.com/cuda/cusolver/index.html#lu_examples
        case CudaSolverType::LU: case CudaSolverType::LU_PIVOT: {
            bool usePivot = cudaSolverType == CudaSolverType::LU_PIVOT;

            // Query working space required by getrf.
            cudaErrorCheck(cusolverDnRgetrf_bufferSize(
                    cusolverHandle, N, N, dLhs, ldLhs, &lwork));
            cudaErrorCheck(hipMalloc((void**)&dWork, sizeof(Real)*lwork));

            // Create data for pivot if user requests pivoting.
            int* dbpiv = nullptr;
            if (usePivot) {
                cudaErrorCheck(hipMalloc((void**)&dbpiv, sizeof(int) * N));
            }

            // LU factorization.
            cudaErrorCheck(cusolverDnRgetrf(
                    cusolverHandle, N, N, dLhs, ldLhs, dWork, dbpiv, dInfo));
            cudaErrorCheck(hipDeviceSynchronize());

            // Check whether the factorization was successful.
            cudaErrorCheck(hipMemcpy(&hInfo, dInfo, sizeof(int), hipMemcpyDeviceToHost));
            if (hInfo != 0) {
                if (usePivot) {
                    std::cerr << "ERROR: CudaSolverType::LU_PIVOT failed!" << std::endl;
                } else {
                    std::cerr << "ERROR: CudaSolverType::LU failed!" << std::endl;
                }
                exit(1);
            }

            // Solve A*l = LU*l = I.
            cudaErrorCheck(cusolverDnRgetrs(
                    cusolverHandle, HIPBLAS_OP_N, N, 1, dLhs, ldLhs, dbpiv, dRhs, ldRhs, dInfo));
            cudaErrorCheck(hipDeviceSynchronize());
            cudaErrorCheck(hipMemcpy(x.data(), dRhs, sizeof(Real) * N, hipMemcpyDeviceToHost));

            if (usePivot && dbpiv) {
                cudaErrorCheck(hipFree(dbpiv));
            }

            break;
        }

        // See: https://docs.nvidia.com/cuda/cusolver/index.html#qr_examples
        case CudaSolverType::QR: {
            const Real one = 1.0f;
            int lwork_geqrf = 0;
            int lwork_ormqr = 0;
            Real* dTau = nullptr;

            cudaErrorCheck(hipMalloc((void**)&dTau, sizeof(Real) * N));

            // Query working space required by geqrf and ormqr.
            cudaErrorCheck(cusolverDnRgeqrf_bufferSize(
                    cusolverHandle, lhsM, lhsN, dLhs, ldLhs, &lwork_geqrf));
            cudaErrorCheck(cusolverDnRormqr_bufferSize(
                    cusolverHandle, HIPBLAS_SIDE_LEFT, HIPBLAS_OP_T,
                    lhsM, 1, lhsN, dLhs, ldLhs, dTau, dRhs, ldRhs, &lwork_ormqr));
            lwork = std::max(lwork_geqrf, lwork_ormqr);
            cudaErrorCheck(hipMalloc((void**)&dWork, sizeof(Real)*lwork));

            // Compute the QR factorization.
            cudaErrorCheck(cusolverDnRgeqrf(
                    cusolverHandle, lhsM, lhsN, dLhs, ldLhs, dTau, dWork, lwork, dInfo));
            cudaErrorCheck(hipDeviceSynchronize());

            // Check whether the factorization was successful.
            cudaErrorCheck(hipMemcpy(&hInfo, dInfo, sizeof(int), hipMemcpyDeviceToHost));
            if (hInfo != 0) {
                std::cerr << "ERROR: CudaSolverType::QR failed!" << std::endl;
                exit(1);
            }

            // Compute Q^T*I.
            cudaErrorCheck(cusolverDnRormqr(
                    cusolverHandle, HIPBLAS_SIDE_LEFT, HIPBLAS_OP_T,
                    lhsM, 1, lhsN, dLhs, ldLhs, dTau, dRhs, ldRhs, dWork, lwork, dInfo));
            cudaErrorCheck(hipDeviceSynchronize());

            // Check whether the factorization was successful.
            cudaErrorCheck(hipMemcpy(&hInfo, dInfo, sizeof(int), hipMemcpyDeviceToHost));
            if (hInfo != 0) {
                std::cerr << "ERROR: CudaSolverType::QR failed!" << std::endl;
                exit(1);
            }

            // Solve R*l = Q^T*I (i.e., l = R \ Q^T*I).
            cudaErrorCheck(cublasRtrsm(
                    cublasHandle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT,
                    lhsN, 1, &one, dLhs, ldLhs, dRhs, ldRhs));
            cudaErrorCheck(hipDeviceSynchronize());
            cudaErrorCheck(hipMemcpy(x.data(), dRhs, sizeof(Real) * N, hipMemcpyDeviceToHost));

            if (dTau) {
                cudaErrorCheck(hipFree(dTau));
            }

            break;
        }

        // https://docs.nvidia.com/cuda/cusolver/index.html#chol_examples
        case CudaSolverType::CHOL: {
            const hipblasFillMode_t fillMode = HIPBLAS_FILL_MODE_LOWER;

            // Query working space required by potrf.
            cudaErrorCheck(cusolverDnRpotrf_bufferSize(
                    cusolverHandle, fillMode, N, dLhs, ldLhs, &lwork));
            cudaErrorCheck(hipMalloc((void**)&dWork, sizeof(Real)*lwork));

            // Cholesky factorization.
            cudaErrorCheck(cusolverDnRpotrf(
                    cusolverHandle, fillMode, N, dLhs, ldLhs, dWork, lwork, dInfo));
            cudaErrorCheck(hipDeviceSynchronize());

            // Check whether the factorization was successful.
            cudaErrorCheck(hipMemcpy(&hInfo, dInfo, sizeof(int), hipMemcpyDeviceToHost));
            if (hInfo != 0) {
                std::cerr << "ERROR: CudaSolverType::CHOL failed!" << std::endl;
                exit(1);
            }

            // Solving step.
            cudaErrorCheck(cusolverDnRpotrs(
                    cusolverHandle, fillMode, N, 1, dLhs, ldLhs, dRhs, ldRhs, dInfo));
            cudaErrorCheck(hipDeviceSynchronize());

            // Check whether the solving step was successful.
            cudaErrorCheck(hipMemcpy(&hInfo, dInfo, sizeof(int), hipMemcpyDeviceToHost));
            if (hInfo != 0) {
                std::cerr << "ERROR: CudaSolverType::CHOL failed!" << std::endl;
                exit(1);
            }

            cudaErrorCheck(hipMemcpy(x.data(), dRhs, sizeof(Real) * N, hipMemcpyDeviceToHost));

            break;
        }
    }

    // Free the allocated memory.
    if (dWork) {
        cudaErrorCheck(hipFree(dWork));
    }
    if (dInfo) {
        cudaErrorCheck(hipFree(dInfo));
    }
    if (useNormalEquations) {
        if (dLhs) {
            cudaErrorCheck(hipFree(dLhs));
        }
        if (dRhs) {
            cudaErrorCheck(hipFree(dRhs));
        }
    }
    if (dA) {
        cudaErrorCheck(hipFree(dA));
    }
    if (db) {
        cudaErrorCheck(hipFree(db));
    }
    if (dx) {
        cudaErrorCheck(hipFree(dx));
    }
}

void solveLeastSquaresCudaSparse(
        int m, int n, int nnz, const Real* csrVals, const int* csrRowPtr, const int* csrColInd,
        const Real* b, Eigen::MatrixXr& x) {
    auto* p = new int[n];
    x = Eigen::MatrixXr(n, 1);
    int rank = 0;
    auto minNorm = Real(0);
    hipsparseMatDescr_t matDesc{};
    cudaErrorCheck(hipsparseCreateMatDescr(&matDesc));
    cudaErrorCheck(cusolverSpRcsrlsqvqrHost(
            cusolverSpHandle, m, n, nnz, matDesc, csrVals, csrRowPtr, csrColInd, b,
            Real(1e-5f), &rank, x.data(), p, &minNorm));
    cudaErrorCheck(hipsparseDestroyMatDescr(matDesc));
    delete[] p;

    // TODO
    bool quiet = false; // TODO: Turn off if it works.
    float tol = 1e-6f;
    int maxit = 100;
    float s = 0.0f;
    //cgls::Solve<Real, cgls::CSR>(csrVals, csrRowPtr, csrColInd, m, n, nnz, b, x, s, tol, maxit, quiet);
}
