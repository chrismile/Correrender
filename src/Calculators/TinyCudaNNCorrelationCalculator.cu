#include "hip/hip_runtime.h"
/*
 * BSD 2-Clause License
 *
 * Copyright (c) 2022, Christoph Neuhauser
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * * Redistributions of source code must retain the above copyright notice, this
 *   list of conditions and the following disclaimer.
 *
 * * Redistributions in binary form must reproduce the above copyright notice,
 *   this list of conditions and the following disclaimer in the documentation
 *   and/or other materials provided with the distribution.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#ifdef TEST_INFERENCE_SPEED
#include <hip/hip_runtime_api.h>
#endif

#include <tiny-cuda-nn/evaluator.h>
#include <tiny-cuda-nn/network_with_input_encoding.h>
#include <tiny-cuda-nn/networks/fully_fused_mlp.h>

#include <Math/Math.hpp>
#include <Utils/File/FileLoader.hpp>
#include <Utils/File/Archive.hpp>
#include <Utils/File/FileUtils.hpp>
#include <Graphics/Vulkan/Utils/InteropCuda.hpp>
#include <Graphics/Vulkan/Render/Renderer.hpp>
#include <ImGui/Widgets/PropertyEditor.hpp>

#include "Utils/InternalState.hpp"
#include "Volume/VolumeData.hpp"
#include "MutualInformation.cuh"
#include "TinyCudaNNCorrelationCalculator.hpp"

using precision_t = tcnn::network_precision_t;

struct TinyCudaNNModuleWrapper {
    nlohmann::json configGeneral;
    nlohmann::json configEncoder;
    nlohmann::json configDecoder;
    std::shared_ptr<tcnn::Network<float, precision_t>> networkEncoder;
    std::shared_ptr<tcnn::Evaluator<float, precision_t, precision_t>> evaluatorEncoder;
#if TCNN_HALF_PRECISION
    std::shared_ptr<tcnn::Network<precision_t, precision_t>> networkEncoderHalf;
    std::shared_ptr<tcnn::Evaluator<precision_t, precision_t, precision_t>> evaluatorEncoderHalf;
#endif
    std::shared_ptr<tcnn::Network<precision_t, precision_t>> networkDecoder;
    std::shared_ptr<tcnn::Evaluator<precision_t, precision_t, precision_t>> evaluatorDecoder;
};

struct TinyCudaNNCacheWrapper {
    tcnn::GPUMatrix<float> referenceInput;
#if TCNN_HALF_PRECISION
    tcnn::GPUMatrix<precision_t> referenceInputHalf;
#endif
    tcnn::GPUMatrix<precision_t> referenceEncoded;
    tcnn::GPUMatrix<float> queryInput;
#if TCNN_HALF_PRECISION
    tcnn::GPUMatrix<precision_t> queryInputHalf;
#endif
    tcnn::GPUMatrix<precision_t> queryEncoded;
    tcnn::GPUMatrix<precision_t> symmetrizedReferenceInput;
    tcnn::GPUMatrix<precision_t> symmetrizedQueryInput;
    tcnn::GPUMatrix<precision_t> referenceDecoded;
    tcnn::GPUMatrix<precision_t> queryDecoded;
    AuxiliaryMemoryToken auxMemoryToken{};
};

TinyCudaNNCorrelationCalculator::TinyCudaNNCorrelationCalculator(sgl::vk::Renderer* renderer)
        : DeepLearningCudaCorrelationCalculator("tiny-cuda-nn", "tinyCudaNN", renderer) {
    cacheWrapper = std::make_shared<TinyCudaNNCacheWrapper>();

    hipDevice_t cuDevice = 0;
    bool foundDevice = sgl::vk::getMatchingCudaDevice(renderer->getDevice(), &cuDevice);
    if (foundDevice) {
        hipError_t cuResult;
        int computeCapabilityMajor = 7;
        cuResult = sgl::vk::g_cudaDeviceApiFunctionTable.hipDeviceGetAttribute(
                &computeCapabilityMajor, hipDeviceAttributeComputeCapabilityMajor, cuDevice);
        sgl::vk::checkCUresult(cuResult, "Error in hipDeviceGetAttribute: ");
        deviceSupporsFullyFusedMlp = computeCapabilityMajor >= 7;
    }

    if (!deviceSupporsFullyFusedMlp) {
        networkImplementation = TinyCudaNNNetworkImplementation::CUTLASS_MLP;
    }
}

TinyCudaNNCorrelationCalculator::~TinyCudaNNCorrelationCalculator() {
    if (cacheWrapper->auxMemoryToken) {
        volumeData->popAuxiliaryMemoryDevice(cacheWrapper->auxMemoryToken);
    }
}

void TinyCudaNNCorrelationCalculator::setVolumeData(VolumeData* _volumeData, bool isNewData) {
    DeepLearningCudaCorrelationCalculator::setVolumeData(_volumeData, isNewData);
    if (isNewData) {
        calculatorConstructorUseCount = volumeData->getNewCalculatorUseCount(CalculatorType::TINY_CUDA_NN);
    }
}

void TinyCudaNNCorrelationCalculator::renderGuiImplAdvanced(sgl::PropertyEditor& propertyEditor) {
    DeepLearningCudaCorrelationCalculator::renderGuiImplAdvanced(propertyEditor);
    if (deviceSupporsFullyFusedMlp && propertyEditor.addCombo(
            "Network", (int*)&networkImplementation,
            TINY_CUDA_NN_NETWORK_IMPLEMENTATION_UI_NAMES, IM_ARRAYSIZE(TINY_CUDA_NN_NETWORK_IMPLEMENTATION_UI_NAMES))) {
        if (sgl::FileUtils::get()->exists(modelFilePath) && !sgl::FileUtils::get()->isDirectory(modelFilePath)) {
            loadModelFromFile(modelFilePath);
        }
        dirty = true;
    }
}

void TinyCudaNNCorrelationCalculator::setSettings(const SettingsMap& settings) {
    std::string networkImplementationString;
    if (settings.getValueOpt("network_implementation", networkImplementationString)) {
        for (int i = 0; i < IM_ARRAYSIZE(TINY_CUDA_NN_NETWORK_IMPLEMENTATION_NAMES); i++) {
            if (networkImplementationString == TINY_CUDA_NN_NETWORK_IMPLEMENTATION_NAMES[i]) {
                networkImplementation = TinyCudaNNNetworkImplementation(i);
                break;
            }
        }
        if (!deviceSupporsFullyFusedMlp) {
            networkImplementation = TinyCudaNNNetworkImplementation::CUTLASS_MLP;
        }
        dirty = true;
    }
    DeepLearningCudaCorrelationCalculator::setSettings(settings);
}

void TinyCudaNNCorrelationCalculator::getSettings(SettingsMap& settings) {
    DeepLearningCudaCorrelationCalculator::getSettings(settings);
    settings.addKeyValue(
            "network_implementation", TINY_CUDA_NN_NETWORK_IMPLEMENTATION_NAMES[int(networkImplementation)]);
}

template<class T, class PARAMS_T> static void loadNetwork(
        std::shared_ptr<tcnn::Network<T, PARAMS_T>>& network,
        std::shared_ptr<tcnn::Evaluator<T, PARAMS_T, PARAMS_T>>& evaluator,
        const std::string& modelPath, const nlohmann::json& config, const sgl::ArchiveEntry& entry) {
    auto* header = reinterpret_cast<NetworkParametersHeader*>(entry.bufferData.get());
    uint8_t* paramsData = entry.bufferData.get() + sizeof(NetworkParametersHeader);
    uint32_t numParams = header->numParams;

    size_t sizePerEntry = header->format == NETWORK_PARAMS_FORMAT_FLOAT ? 4 : 2;
    if (numParams * sizePerEntry + sizeof(NetworkParametersHeader) != entry.bufferSize) {
        sgl::Logfile::get()->throwError(
                "Error in loadNetwork: Invalid number of parameters for file size.");
    }

    bool hasInputEncoding = config.find("encoding") != config.end();
    bool isInputEncodingIdentity = false;
    auto encodingOpts = config.value("encoding", nlohmann::json::object());
    auto lossOpts = config.value("loss", nlohmann::json::object());
    auto optimizerOpts = config.value("optimizer", nlohmann::json::object());
    auto networkOpts = config.value("network", nlohmann::json::object());
    if (hasInputEncoding) {
        if (encodingOpts.value("otype", "Identity") == "Identity"
                && encodingOpts.value("scale", 1.0f) == 1.0f
                && encodingOpts.value("offset", 0.0f) == 0.0f) {
            isInputEncodingIdentity = true;
        }
    }

    uint32_t numInputDims = networkOpts["n_input_dims"];
    uint32_t numOutputDims = networkOpts["n_output_dims"];
    //std::shared_ptr<tcnn::Loss<precision_t>> loss{tcnn::create_loss<precision_t>(lossOpts)};
    //std::shared_ptr<tcnn::Optimizer<precision_t>> optimizer{tcnn::create_optimizer<precision_t>(optimizerOpts)};
    if (hasInputEncoding && !isInputEncodingIdentity) {
        std::shared_ptr<tcnn::NetworkWithInputEncoding<precision_t>> networkWithEnc =
                std::make_shared<tcnn::NetworkWithInputEncoding<precision_t>>(
                        numInputDims, numOutputDims, encodingOpts, networkOpts);
        if constexpr (std::is_same<T, float>::value) {
            network = std::static_pointer_cast<tcnn::Network<float, PARAMS_T>>(networkWithEnc);
        }
    } else {
        if constexpr (std::is_same<T, PARAMS_T>::value) {
            network = std::shared_ptr<tcnn::Network<PARAMS_T, PARAMS_T>>(
                    tcnn::create_network<PARAMS_T>(networkOpts));
        }
    }
    //network->set_params();
    evaluator = std::make_shared<tcnn::Evaluator<T, PARAMS_T, PARAMS_T>>(network);

    // Do we need padding because the output width is not a multiple of 16?
    if (network->output_width() != network->padded_output_width() && network->n_params() != numParams) {
        uint32_t numNeurons = networkOpts["n_neurons"];
        uint32_t paddingSize = numNeurons * (network->padded_output_width() - network->output_width());
        size_t numParamsOld = numParams;
        numParams += paddingSize;
        const uint8_t* paramsDataOld = paramsData;
        paramsData = new uint8_t[numParams * sizePerEntry];
        memcpy(paramsData, paramsDataOld, numParamsOld * sizePerEntry);
        memset(paramsData + numParamsOld * sizePerEntry, 0, paddingSize * sizePerEntry);
    }

    if (network->n_params() != numParams) {
        sgl::Logfile::get()->throwError(
                "Error in loadNetwork: Mismatching network parameter count (" + std::to_string(numParams)
                + " vs. " + std::to_string(network->n_params()) + ") for \"" + modelPath + "\".");
    }

#if TCNN_HALF_PRECISION
    if (header->format == NETWORK_PARAMS_FORMAT_FLOAT) {
        evaluator->set_params_full_precision(reinterpret_cast<float*>(paramsData), numParams, false);
    } else {
        evaluator->set_params(reinterpret_cast<precision_t*>(paramsData), numParams, false);
    }
#else
    if (header->format == NETWORK_PARAMS_FORMAT_FLOAT) {
        evaluator->set_params(reinterpret_cast<float*>(paramsData), numParams, false);
    } else {
        sgl::Logfile::get()->throwError(
                "Error in TinyCudaNNCorrelationCalculator::loadNetwork: Half precision build was disabled.");
    }
#endif

    if (network->output_width() != network->padded_output_width() && network->n_params() != numParams) {
        delete[] paramsData;
    }
}

void TinyCudaNNCorrelationCalculator::loadModelFromFile(const std::string& modelPath) {
    moduleWrapper = std::make_shared<TinyCudaNNModuleWrapper>();
    cacheWrapper = std::make_shared<TinyCudaNNCacheWrapper>();

    std::unordered_map<std::string, sgl::ArchiveEntry> archiveFiles;
    sgl::ArchiveFileLoadReturnType retVal = sgl::loadAllFilesFromArchive(modelPath, archiveFiles, true);
    if (retVal != sgl::ArchiveFileLoadReturnType::ARCHIVE_FILE_LOAD_SUCCESSFUL) {
        sgl::Logfile::get()->writeError(
                "Error in TinyCudaNNCorrelationCalculator::loadModelFromFile: Could not load data from model \""
                + modelPath + "\".");
        return;
    }

    // A global configuration file is optional.
    auto itConfig = archiveFiles.find("config.json");
    if (itConfig != archiveFiles.end()) {
        const auto& entry = itConfig->second;
        moduleWrapper->configGeneral = nlohmann::json::parse(std::string(
                reinterpret_cast<char*>(entry.bufferData.get()), entry.bufferSize));

        auto symmetrizerTypeName = moduleWrapper->configGeneral.value(
                "symmetrizer_type", SYMMETRIZER_TYPE_SHORT_NAMES[0]);
        bool foundSymmetrizerType = false;
        for (int i = 0; i < IM_ARRAYSIZE(SYMMETRIZER_TYPE_SHORT_NAMES); i++) {
            if (SYMMETRIZER_TYPE_SHORT_NAMES[i] == symmetrizerTypeName) {
                symmetrizerType = SymmetrizerType(i);
                foundSymmetrizerType = true;
                break;
            }
        }
        if (!foundSymmetrizerType) {
            sgl::Logfile::get()->writeError(
                    "Error in TinyCudaNNCorrelationCalculator::loadModelFromFile: Invalid symmetrizer type \""
                    + symmetrizerTypeName + "\".");
            return;
        }

        auto networkTypeName = moduleWrapper->configGeneral.value(
                "network_type", NETWORK_TYPE_SHORT_NAMES[0]);
        bool foundNetworkType = false;
        for (int i = 0; i < IM_ARRAYSIZE(NETWORK_TYPE_SHORT_NAMES); i++) {
            if (NETWORK_TYPE_SHORT_NAMES[i] == networkTypeName) {
                networkType = NetworkType(i);
                foundNetworkType = true;
                break;
            }
        }
        if (!foundNetworkType && networkTypeName == "MINE_SRN") {
            networkType = NetworkType::SRN_MINE;
            foundNetworkType = true;
        }
        if (!foundNetworkType) {
            sgl::Logfile::get()->writeError(
                    "Error in TinyCudaNNCorrelationCalculator::loadModelFromFile: Invalid network type \""
                    + networkTypeName + "\".");
            return;
        }

        isMutualInformationData = moduleWrapper->configGeneral.value("is_mutual_information", true);
    }

    // Encoder and decoder configuration files are mandatory.
    auto itConfigEncoder = archiveFiles.find("config_encoder.json");
    auto itConfigDecoder = archiveFiles.find("config_decoder.json");
    if (itConfigEncoder != archiveFiles.end() && itConfigDecoder != archiveFiles.end()) {
        const auto& entryEncoder = itConfigEncoder->second;
        const auto& entryDecoder = itConfigDecoder->second;
        moduleWrapper->configEncoder = nlohmann::json::parse(std::string(
                reinterpret_cast<char*>(entryEncoder.bufferData.get()), entryEncoder.bufferSize));
        moduleWrapper->configDecoder = nlohmann::json::parse(std::string(
                reinterpret_cast<char*>(entryDecoder.bufferData.get()), entryDecoder.bufferSize));
    } else {
        sgl::Logfile::get()->writeError(
                "Error in TinyCudaNNCorrelationCalculator::loadModelFromFile: Could not load encoder or decoder "
                "configuration from model \"" + modelPath + "\".");
        return;
    }

    isInputEncodingIdentity = false;
    bool hasInputEncoding = moduleWrapper->configEncoder.find("encoding") != moduleWrapper->configEncoder.end();
    auto encodingOpts = moduleWrapper->configEncoder.value("encoding", nlohmann::json::object());
    if (hasInputEncoding) {
        if (encodingOpts.value("otype", "Identity") == "Identity"
                && encodingOpts.value("scale", 1.0f) == 1.0f
                && encodingOpts.value("offset", 0.0f) == 0.0f) {
            isInputEncodingIdentity = true;
        }
    }

    // Set input/output layer configurations for both networks.
    auto encoderNetworkOpts = moduleWrapper->configEncoder.value("network", nlohmann::json::object());
    auto decoderNetworkOpts = moduleWrapper->configDecoder.value("network", nlohmann::json::object());
    // mlp_fused_forward needs multiple of 16 for number of input layers.
    int numInputLayers = 16;
    if (!isInputEncodingIdentity) {
        if (networkType == NetworkType::MINE) {
            numInputLayers = 4;
        } else {
            numInputLayers = 3;
        }
    }
    moduleWrapper->configEncoder["network"]["n_input_dims"] = numInputLayers;
    moduleWrapper->configDecoder["network"]["n_output_dims"] = 1;
    if (encoderNetworkOpts.find("n_output_dims") == encoderNetworkOpts.end()) {
        moduleWrapper->configEncoder["network"]["n_output_dims"] = moduleWrapper->configEncoder["network"]["n_neurons"];
    }
    uint32_t symmetrizerFactor = symmetrizerType == SymmetrizerType::AddDiff ? 2 : 1;
    if (decoderNetworkOpts.find("n_input_dims") == decoderNetworkOpts.end()) {
        uint32_t encoderOutputDims = moduleWrapper->configEncoder["network"].value("n_output_dims", 0);
        moduleWrapper->configDecoder["network"]["n_input_dims"] = encoderOutputDims * symmetrizerFactor;
    }

    const char* networkTypeName = TINY_CUDA_NN_NETWORK_IMPLEMENTATION_NAMES[int(networkImplementation)];
    moduleWrapper->configEncoder["network"]["otype"] = networkTypeName;
    moduleWrapper->configDecoder["network"]["otype"] = networkTypeName;

    auto itNetworkEncoder = archiveFiles.find("network_encoder.bin");
    auto itNetworkDecoder = archiveFiles.find("network_decoder.bin");
    if (itNetworkEncoder == archiveFiles.end()) {
        sgl::Logfile::get()->writeError(
                "Error in TinyCudaNNCorrelationCalculator::loadModelFromFile: Missing network_encoder.bin in file \""
                + modelPath + "\".");
        return;
    }
    if (itNetworkDecoder == archiveFiles.end()) {
        sgl::Logfile::get()->writeError(
                "Error in TinyCudaNNCorrelationCalculator::loadModelFromFile: Missing network_decoder.bin in file \""
                + modelPath + "\".");
        return;
    }
    moduleWrapper->networkEncoder = {};
    moduleWrapper->evaluatorEncoder = {};
#if TCNN_HALF_PRECISION
    moduleWrapper->networkEncoderHalf = {};
    moduleWrapper->evaluatorEncoderHalf = {};
#endif
    moduleWrapper->networkDecoder = {};
    moduleWrapper->evaluatorDecoder = {};
#if TCNN_HALF_PRECISION
    if (hasInputEncoding && !isInputEncodingIdentity) {
#endif
        loadNetwork(
                moduleWrapper->networkEncoder, moduleWrapper->evaluatorEncoder, modelPath,
                moduleWrapper->configEncoder, itNetworkEncoder->second);
#if TCNN_HALF_PRECISION
    } else {
        loadNetwork(
                moduleWrapper->networkEncoderHalf, moduleWrapper->evaluatorEncoderHalf, modelPath,
                moduleWrapper->configEncoder, itNetworkEncoder->second);
    }
#endif
    loadNetwork(
            moduleWrapper->networkDecoder, moduleWrapper->evaluatorDecoder, modelPath,
            moduleWrapper->configDecoder, itNetworkDecoder->second);

    // numLayersOutEncoder == numLayersInDecoder when symmetrizer is sum operation.
#if TCNN_HALF_PRECISION
    if (moduleWrapper->networkEncoderHalf) {
        numLayersInEncoder = uint32_t(moduleWrapper->networkEncoderHalf->input_width());
        numLayersOutEncoder = uint32_t(moduleWrapper->networkEncoderHalf->padded_output_width());
    } else {
        numLayersInEncoder = uint32_t(moduleWrapper->networkEncoder->input_width());
        numLayersOutEncoder = uint32_t(moduleWrapper->networkEncoder->padded_output_width());
    }
#else
    numLayersInEncoder = uint32_t(moduleWrapper->networkEncoder->input_width());
    numLayersOutEncoder = uint32_t(moduleWrapper->networkEncoder->padded_output_width());
#endif

    numLayersInDecoder = uint32_t(moduleWrapper->networkDecoder->input_width());
#if TCNN_HALF_PRECISION
    numLayersOutDecoder = uint32_t(moduleWrapper->networkDecoder->padded_output_width());
#else
    // tcnn::DifferentiableObject<T,PARAMS_T,COMPUTE_T>::inference checks output.m() == output_width().
    // For some reason, there is an incompatibility for the CutlassMLP class.
    numLayersOutDecoder = uint32_t(moduleWrapper->networkDecoder->output_width());
#endif

    if (numLayersOutEncoder * symmetrizerFactor != numLayersInDecoder) {
        sgl::Logfile::get()->throwError(
                "Error in TinyCudaNNCorrelationCalculator::loadModelFromFile: Mismatch between encoder output and "
                "decoder input dimensions.");
    }

    cacheNeedsRecreate = true;
}

void TinyCudaNNCorrelationCalculator::recreateCache(int batchSize) {
    int cs = networkType == NetworkType::MINE ? getCorrelationMemberCount() : 1;

    cacheWrapper->referenceInput = tcnn::GPUMatrix<float>();
#if TCNN_HALF_PRECISION
    cacheWrapper->referenceInputHalf = tcnn::GPUMatrix<precision_t>();
#endif
    cacheWrapper->referenceEncoded = tcnn::GPUMatrix<precision_t>();
    cacheWrapper->queryInput = tcnn::GPUMatrix<float>();
#if TCNN_HALF_PRECISION
    cacheWrapper->queryInputHalf = tcnn::GPUMatrix<precision_t>();
#endif
    cacheWrapper->queryEncoded = tcnn::GPUMatrix<precision_t>();
    cacheWrapper->symmetrizedReferenceInput = tcnn::GPUMatrix<precision_t>();
    cacheWrapper->symmetrizedQueryInput = tcnn::GPUMatrix<precision_t>();
    cacheWrapper->referenceDecoded = tcnn::GPUMatrix<precision_t>();
    cacheWrapper->queryDecoded = tcnn::GPUMatrix<precision_t>();
    if (cacheWrapper->auxMemoryToken) {
        volumeData->popAuxiliaryMemoryDevice(cacheWrapper->auxMemoryToken);
    }

    // mlp_fused_forward needs multiple of 16 for number of input layers.
    uint32_t numInputLayers = 16;
    if (!isInputEncodingIdentity) {
        if (networkType == NetworkType::MINE) {
            numInputLayers = 4;
        } else {
            numInputLayers = 3;
        }
    }
    uint32_t referenceInputBatchSize =
            sgl::uiceil(uint32_t(cs), tcnn::batch_size_granularity) * tcnn::batch_size_granularity;
#if TCNN_HALF_PRECISION
    if (moduleWrapper->networkEncoderHalf) {
        cacheWrapper->referenceInputHalf = tcnn::GPUMatrix<precision_t>(numInputLayers, referenceInputBatchSize);
        cacheWrapper->queryInputHalf = tcnn::GPUMatrix<precision_t>(numInputLayers, uint32_t(cs) * batchSize);
    }
#endif
    cacheWrapper->referenceInput = tcnn::GPUMatrix<float>(numInputLayers, referenceInputBatchSize);
    cacheWrapper->queryInput = tcnn::GPUMatrix<float>(numInputLayers, uint32_t(cs) * batchSize);
    cacheWrapper->referenceEncoded = tcnn::GPUMatrix<precision_t>(numLayersOutEncoder, referenceInputBatchSize);
    cacheWrapper->queryEncoded = tcnn::GPUMatrix<precision_t>(numLayersOutEncoder, uint32_t(cs) * batchSize);
    cacheWrapper->symmetrizedReferenceInput = tcnn::GPUMatrix<precision_t>(numLayersInDecoder, uint32_t(cs) * batchSize);
    cacheWrapper->symmetrizedQueryInput = tcnn::GPUMatrix<precision_t>(numLayersInDecoder, uint32_t(cs) * batchSize);
    cacheWrapper->referenceDecoded = tcnn::GPUMatrix<precision_t>(numLayersOutDecoder, uint32_t(cs) * batchSize);
    cacheWrapper->queryDecoded = tcnn::GPUMatrix<precision_t>(numLayersOutDecoder, uint32_t(cs) * batchSize);

    size_t auxBuffersSizeInBytes = 0;
#if TCNN_HALF_PRECISION
    if (moduleWrapper->networkEncoderHalf) {
        auxBuffersSizeInBytes += size_t(cacheWrapper->referenceInputHalf.n_bytes());
        auxBuffersSizeInBytes += size_t(cacheWrapper->queryInputHalf.n_bytes());
    }
#endif
    auxBuffersSizeInBytes += size_t(cacheWrapper->referenceInput.n_bytes());
    auxBuffersSizeInBytes += size_t(cacheWrapper->queryInput.n_bytes());
    auxBuffersSizeInBytes += size_t(cacheWrapper->referenceEncoded.n_bytes());
    auxBuffersSizeInBytes += size_t(cacheWrapper->queryEncoded.n_bytes());
    auxBuffersSizeInBytes += size_t(cacheWrapper->symmetrizedReferenceInput.n_bytes());
    auxBuffersSizeInBytes += size_t(cacheWrapper->symmetrizedQueryInput.n_bytes());
    auxBuffersSizeInBytes += size_t(cacheWrapper->referenceDecoded.n_bytes());
    auxBuffersSizeInBytes += size_t(cacheWrapper->queryDecoded.n_bytes());
    cacheWrapper->auxMemoryToken = volumeData->pushAuxiliaryMemoryDevice(auxBuffersSizeInBytes);
}

hipDeviceptr_t TinyCudaNNCorrelationCalculator::getReferenceInputPointer()  {
    return reinterpret_cast<hipDeviceptr_t>(cacheWrapper->referenceInput.data());
}

hipDeviceptr_t TinyCudaNNCorrelationCalculator::getQueryInputPointer()  {
    return reinterpret_cast<hipDeviceptr_t>(cacheWrapper->queryInput.data());
}

void TinyCudaNNCorrelationCalculator::runInferenceReference() {
#if TCNN_HALF_PRECISION
    if (moduleWrapper->networkEncoderHalf) {
        uint32_t arraySize = cacheWrapper->referenceInputHalf.n() * cacheWrapper->referenceInputHalf.m();
        convertFloatToHalfArray<<<sgl::uiceil(arraySize, 256), 256, 0, stream>>>(
                cacheWrapper->referenceInputHalf.data(), cacheWrapper->referenceInput.data(), arraySize);
        moduleWrapper->networkEncoderHalf->inference_mixed_precision(
                stream, cacheWrapper->referenceInputHalf, cacheWrapper->referenceEncoded);
    } else {
        moduleWrapper->networkEncoder->inference_mixed_precision(
                stream, cacheWrapper->referenceInput, cacheWrapper->referenceEncoded);
    }
#else
    moduleWrapper->networkEncoder->inference(
                stream, cacheWrapper->referenceInput, cacheWrapper->referenceEncoded);
#endif

    /*float* refInput = new float[3];
    sgl::vk::checkCUresult(sgl::vk::g_cudaDeviceApiFunctionTable.hipMemcpyDtoHAsync(
            refInput, (hipDeviceptr_t)cacheWrapper->referenceInput.data(),
            sizeof(float) * 3, stream), "Error in hipMemcpyDtoHAsync: ");
    sgl::vk::checkCUresult(sgl::vk::g_cudaDeviceApiFunctionTable.hipStreamSynchronize(
            stream), "Error in hipStreamSynchronize: ");
    std::cout << "refInput:" << std::endl;
    for (int i = 0; i < 3; i++) {
        std::cout << float(refInput[i]);
        if (i != 2) {
            std::cout << ", ";
        } else {
            std::cout << std::endl;
        }
    }
    std::cout << std::endl;
    delete[] refInput;*/

    /*int copySize = batchSize * 40;
    int testSize = 160;
    __half* dataHalf = new __half[copySize];
    dataHalf[0] = 1000.0f;
    sgl::vk::checkCUresult(sgl::vk::g_cudaDeviceApiFunctionTable.hipMemcpyDtoHAsync(
            dataHalf, (hipDeviceptr_t)cacheWrapper->queryInputHalf.data(),
            sizeof(__half) * copySize, stream), "Error in hipMemcpyDtoHAsync: ");
    sgl::vk::checkCUresult(sgl::vk::g_cudaDeviceApiFunctionTable.hipStreamSynchronize(
            stream), "Error in hipStreamSynchronize: ");
    std::cout << "queryInputHalf:" << std::endl;
    for (int i = 0; i < testSize; i++) {
        std::cout << float(dataHalf[i * numLayersInEncoder * uint32_t(es)]);
        if (i != testSize - 1) {
            std::cout << ", ";
        }
        if (i % 20 == 19 && i != 0) {
            std::cout << std::endl;
        }
    }
    std::cout << std::endl;
    delete[] dataHalf;*/

    /*int sizeDataEnc = 4 * 64;
    auto* dataEnc = new __half[sizeDataEnc];
    hipMemcpy(dataEnc, cacheWrapper->referenceEncoded.data(), sizeDataEnc * sizeof(__half), hipMemcpyDeviceToHost);
    std::cout << "dataEnc:" << std::endl;
    for (int i = 0; i < sizeDataEnc; i++) {
        std::cout << float(dataEnc[i]);
        if (i != sizeDataEnc - 1) {
            std::cout << ", ";
        }
        if (i % 16 == 15 && i != 0) {
            std::cout << std::endl;
        }
    }
    std::cout << std::endl;
    delete[] dataEnc;*/
}

void TinyCudaNNCorrelationCalculator::runInferenceBatch(uint32_t batchOffset, uint32_t batchSize)  {
    int cs = getCorrelationMemberCount();

#if TCNN_HALF_PRECISION
    if (moduleWrapper->networkEncoderHalf) {
        uint32_t arraySize = cacheWrapper->queryInputHalf.n() * cacheWrapper->queryInputHalf.m();
        convertFloatToHalfArray<<<sgl::uiceil(arraySize, 256), 256, 0, stream>>>(
                cacheWrapper->queryInputHalf.data(), cacheWrapper->queryInput.data(), arraySize);
        moduleWrapper->networkEncoderHalf->inference_mixed_precision(
                stream, cacheWrapper->queryInputHalf, cacheWrapper->queryEncoded);
    } else {
        moduleWrapper->networkEncoder->inference_mixed_precision(
                stream, cacheWrapper->queryInput, cacheWrapper->queryEncoded);
    }
#else
    moduleWrapper->networkEncoder->inference(
                stream, cacheWrapper->queryInput, cacheWrapper->queryEncoded);
#endif

    /*int copySize = 3 * 800;
    int testSize = 160;
    float* dataFloat = new float[copySize];
    dataFloat[0] = 1000.0f;
    sgl::vk::checkCUresult(sgl::vk::g_cudaDeviceApiFunctionTable.hipMemcpyDtoHAsync(
            dataFloat, (hipDeviceptr_t)cacheWrapper->queryInput.data(),
            sizeof(float) * copySize, stream), "Error in hipMemcpyDtoHAsync: ");
    sgl::vk::checkCUresult(sgl::vk::g_cudaDeviceApiFunctionTable.hipStreamSynchronize(
            stream), "Error in hipStreamSynchronize: ");
    std::cout << "queryInput:" << std::endl;
    for (int i = 0; i < testSize; i++) {
        std::cout << dataFloat[i];
        if (i != testSize - 1) {
            std::cout << ", ";
        }
        if (i % 20 == 19 && i != 0) {
            std::cout << std::endl;
        }
    }
    std::cout << std::endl;
    delete[] dataFloat;*/

    /*int copySize = batchSize * 40;
    int testSize = 160;
    float* dataFloat = new float[copySize];
    dataFloat[0] = 1000.0f;
    sgl::vk::checkCUresult(sgl::vk::g_cudaDeviceApiFunctionTable.hipMemcpyDtoHAsync(
            dataFloat, (hipDeviceptr_t)cacheWrapper->queryInput.data(),
            sizeof(float) * copySize, stream), "Error in hipMemcpyDtoHAsync: ");
    sgl::vk::checkCUresult(sgl::vk::g_cudaDeviceApiFunctionTable.hipStreamSynchronize(
            stream), "Error in hipStreamSynchronize: ");
    std::cout << "queryInput:" << std::endl;
    for (int i = 0; i < testSize; i++) {
        //std::cout << dataFloat[i * numLayersInEncoder * uint32_t(es)];
        std::cout << dataFloat[i];
        if (i != testSize - 1) {
            std::cout << ", ";
        }
        if (i % 20 == 19 && i != 0) {
            std::cout << std::endl;
        }
    }
    std::cout << std::endl;
    delete[] dataFloat;
    __half* dataHalf = new __half[copySize];
    dataHalf = new __half[copySize];
    dataHalf[0] = 1000.0f;
    sgl::vk::checkCUresult(sgl::vk::g_cudaDeviceApiFunctionTable.hipMemcpyDtoHAsync(
            dataHalf, (hipDeviceptr_t)cacheWrapper->queryEncoded.data(),
            sizeof(__half) * copySize, stream), "Error in hipMemcpyDtoHAsync: ");
    sgl::vk::checkCUresult(sgl::vk::g_cudaDeviceApiFunctionTable.hipStreamSynchronize(
            stream), "Error in hipStreamSynchronize: ");
    std::cout << "queryEncoded:" << std::endl;
    for (int i = 0; i < testSize; i++) {
        //std::cout << float(dataHalf[i * numLayersOutEncoder * uint32_t(es)]);
        std::cout << float(dataHalf[i]);
        if (i != testSize - 1) {
            std::cout << ", ";
        }
        if (i % 20 == 19 && i != 0) {
            std::cout << std::endl;
        }
    }
    std::cout << std::endl;
    std::cout << "queryEncoded(2):" << std::endl;
    for (int i = 0; i < testSize; i++) {
        //std::cout << float(dataHalf[i * numLayersOutEncoder * uint32_t(es) + numLayersOutEncoder]);
        std::cout << float(dataHalf[i]);
        if (i != testSize - 1) {
            std::cout << ", ";
        }
        if (i % 20 == 19 && i != 0) {
            std::cout << std::endl;
        }
    }
    std::cout << std::endl;
    delete[] dataHalf;*/

    /*tcnn::FullyFusedMLP<precision_t, 128>* fmlp = (tcnn::FullyFusedMLP<precision_t, 128>*)moduleWrapper->networkEncoderHalf.get();
    const auto& tensor = fmlp->weight_matrix_at(tcnn::WeightUsage::Inference, 1);

    sgl::vk::checkCUresult(sgl::vk::g_cudaDeviceApiFunctionTable.hipMemsetD16Async(
            (hipDeviceptr_t)tensor.data(), 0, 1, stream), "Error in hipMemsetD16Async: ");*/

    /*sgl::vk::checkCUresult(sgl::vk::g_cudaDeviceApiFunctionTable.cuMemcpyAsync(
            (hipDeviceptr_t)cacheWrapper->queryEncodedPermuted.data(),
            (hipDeviceptr_t)cacheWrapper->queryEncoded.data(),
            cacheWrapper->queryEncoded.n() * cacheWrapper->queryEncoded.m(),
            stream), "Error in cuMemcpyAsync: ");*/

    if (networkType == NetworkType::MINE) {
        uint32_t* permutationIndicesBuffer = reinterpret_cast<uint32_t*>(permutationIndicesBufferCu);
        generateRandomPermutations<<<sgl::uiceil(batchSize, 256), 256, 0, stream>>>(
                permutationIndicesBuffer, uint32_t(cs), batchOffset);
        //randomShuffleFisherYates<<<sgl::uiceil(batchSize, 256), 256, 0, stream>>>(
        //        cacheWrapper->queryEncodedPermuted.data(), cacheWrapper->queryEncoded.data(),
        //        permutationIndicesBuffer, uint32_t(es), numLayersOutEncoder);
        symmetrizer(
                cacheWrapper->referenceEncoded.data(), cacheWrapper->queryEncoded.data(),
                cacheWrapper->symmetrizedReferenceInput.data(), cacheWrapper->symmetrizedQueryInput.data(),
                permutationIndicesBuffer, batchSize, uint32_t(cs), numLayersOutEncoder, symmetrizerType, stream);
    } else {
        symmetrizerSrn(
                cacheWrapper->referenceEncoded.data(), cacheWrapper->queryEncoded.data(),
                cacheWrapper->symmetrizedQueryInput.data(),
                batchSize, numLayersOutEncoder, symmetrizerType, stream);
    }

    /*int testSize = 10 * es;
    auto* dataUint32 = new uint32_t[batchSize * es];
    dataUint32[0] = 1000;
    sgl::vk::checkCUresult(sgl::vk::g_cudaDeviceApiFunctionTable.hipMemcpyDtoHAsync(
            dataUint32, (hipDeviceptr_t)permutationIndicesBuffer,
            sizeof(uint32_t) * batchSize * es, stream), "Error in hipMemcpyDtoHAsync: ");
    sgl::vk::checkCUresult(sgl::vk::g_cudaDeviceApiFunctionTable.hipStreamSynchronize(
            stream), "Error in hipStreamSynchronize: ");
    std::cout << "permutationIndicesBuffer:" << std::endl;
    for (int i = 0; i < testSize; i++) {
        std::cout << dataUint32[i];
        if (i != testSize - 1) {
            std::cout << ", ";
        }
        if (i % es == (es - 1) && i != 0) {
            std::cout << std::endl;
        }
    }
    std::cout << std::endl;
    delete[] dataUint32;*/

    /*dataHalf = new __half[copySize];
    dataHalf[0] = 1000.0f;
    sgl::vk::checkCUresult(sgl::vk::g_cudaDeviceApiFunctionTable.hipMemcpyDtoHAsync(
            dataHalf, (hipDeviceptr_t)cacheWrapper->queryEncodedPermuted.data(),
            sizeof(__half) * copySize, stream), "Error in hipMemcpyDtoHAsync: ");
    sgl::vk::checkCUresult(sgl::vk::g_cudaDeviceApiFunctionTable.hipStreamSynchronize(
            stream), "Error in hipStreamSynchronize: ");
    std::cout << "queryEncodedPermuted:" << std::endl;
    for (int i = 0; i < testSize; i++) {
        std::cout << float(dataHalf[i * numLayersOutEncoder * uint32_t(es)]);
        if (i != testSize - 1) {
            std::cout << ", ";
        }
        if (i % 20 == 19 && i != 0) {
            std::cout << std::endl;
        }
    }
    std::cout << std::endl;
    delete[] dataHalf;*/

    /*symmetrizerAdd<<<sgl::uiceil(batchSize * uint32_t(es) * numLayersOutEncoder, 256), 256, 0, stream>>>(
            cacheWrapper->referenceEncoded.data(), cacheWrapper->queryEncoded.data(),
            cacheWrapper->symmetrizedReferenceInput.data(), uint32_t(es), numLayersOutEncoder);
    //symmetrizerAdd<<<sgl::uiceil(batchSize * uint32_t(es) * numLayersOutEncoder, 256), 256, 0, stream>>>(
    //        cacheWrapper->referenceEncoded.data(), cacheWrapper->queryEncodedPermuted.data(),
    //        cacheWrapper->symmetrizedQueryInput.data(), uint32_t(es), numLayersOutEncoder);
    symmetrizerAddPermuted<<<sgl::uiceil(batchSize * uint32_t(es) * numLayersOutEncoder, 256), 256, 0, stream>>>(
            cacheWrapper->referenceEncoded.data(), cacheWrapper->queryEncoded.data(),
            cacheWrapper->symmetrizedQueryInput.data(), permutationIndicesBuffer,
            uint32_t(es), numLayersOutEncoder);*/

    /*dataHalf = new __half[copySize];
    dataHalf[0] = 1000.0f;
    sgl::vk::checkCUresult(sgl::vk::g_cudaDeviceApiFunctionTable.hipMemcpyDtoHAsync(
            dataHalf, (hipDeviceptr_t)cacheWrapper->symmetrizedReferenceInput.data(),
            sizeof(__half) * copySize, stream), "Error in hipMemcpyDtoHAsync: ");
    sgl::vk::checkCUresult(sgl::vk::g_cudaDeviceApiFunctionTable.hipStreamSynchronize(
            stream), "Error in hipStreamSynchronize: ");
    std::cout << "symmetrizedReferenceInput:" << std::endl;
    for (int i = 0; i < testSize; i++) {
        std::cout << float(dataHalf[i]);
        if (i != testSize - 1) {
            std::cout << ", ";
        }
        if (i % 20 == 19 && i != 0) {
            std::cout << std::endl;
        }
    }
    std::cout << std::endl;
    sgl::vk::checkCUresult(sgl::vk::g_cudaDeviceApiFunctionTable.hipMemcpyDtoHAsync(
            dataHalf, (hipDeviceptr_t)cacheWrapper->symmetrizedQueryInput.data(),
            sizeof(__half) * copySize, stream), "Error in hipMemcpyDtoHAsync: ");
    sgl::vk::checkCUresult(sgl::vk::g_cudaDeviceApiFunctionTable.hipStreamSynchronize(
            stream), "Error in hipStreamSynchronize: ");
    std::cout << "symmetrizedQueryInput:" << std::endl;
    for (int i = 0; i < testSize; i++) {
        std::cout << float(dataHalf[i]);
        if (i != testSize - 1) {
            std::cout << ", ";
        }
        if (i % 20 == 19 && i != 0) {
            std::cout << std::endl;
        }
    }
    std::cout << std::endl;
    delete[] dataHalf;*/

#if TCNN_HALF_PRECISION
    if (networkType == NetworkType::MINE) {
        moduleWrapper->networkDecoder->inference_mixed_precision(
                stream, cacheWrapper->symmetrizedReferenceInput, cacheWrapper->referenceDecoded);
    }
    moduleWrapper->networkDecoder->inference_mixed_precision(
            stream, cacheWrapper->symmetrizedQueryInput, cacheWrapper->queryDecoded);
#else
    if (networkType == NetworkType::MINE) {
        moduleWrapper->networkDecoder->inference(
                stream, cacheWrapper->symmetrizedReferenceInput, cacheWrapper->referenceDecoded);
    }
    moduleWrapper->networkDecoder->inference(
            stream, cacheWrapper->symmetrizedQueryInput, cacheWrapper->queryDecoded);
#endif

    /*dataHalf = new __half[copySize];
    dataHalf[0] = 1000.0f;
    sgl::vk::checkCUresult(sgl::vk::g_cudaDeviceApiFunctionTable.hipMemcpyDtoHAsync(
            dataHalf, (hipDeviceptr_t)cacheWrapper->queryDecoded.data(),
            sizeof(__half) * copySize, stream), "Error in hipMemcpyDtoHAsync: ");
    sgl::vk::checkCUresult(sgl::vk::g_cudaDeviceApiFunctionTable.hipStreamSynchronize(
            stream), "Error in hipStreamSynchronize: ");
    std::cout << "queryDecoded:" << std::endl;
    for (int i = 0; i < testSize; i++) {
        std::cout << float(dataHalf[i * numLayersOutDecoder * uint32_t(es)]);
        if (i != testSize - 1) {
            std::cout << ", ";
        }
        if (i % 20 == 19 && i != 0) {
            std::cout << std::endl;
        }
    }
    std::cout << std::endl;
    sgl::vk::checkCUresult(sgl::vk::g_cudaDeviceApiFunctionTable.hipMemcpyDtoHAsync(
            dataHalf, (hipDeviceptr_t)cacheWrapper->referenceDecoded.data(),
            sizeof(__half) * copySize, stream), "Error in hipMemcpyDtoHAsync: ");
    sgl::vk::checkCUresult(sgl::vk::g_cudaDeviceApiFunctionTable.hipStreamSynchronize(
            stream), "Error in hipStreamSynchronize: ");
    std::cout << "referenceDecoded:" << std::endl;
    for (int i = 0; i < testSize; i++) {
        std::cout << float(dataHalf[i * numLayersOutDecoder * uint32_t(es)]);
        if (i != testSize - 1) {
            std::cout << ", ";
        }
        if (i % 20 == 19 && i != 0) {
            std::cout << std::endl;
        }
    }
    std::cout << std::endl;
    delete[] dataHalf;*/

    float* miOutput = reinterpret_cast<float*>(outputImageBufferCu) + batchOffset;
    if (networkType == NetworkType::MINE) {
        combineDecoderOutput<<<sgl::uiceil(batchSize, 256), 256, 0, stream>>>(
                cacheWrapper->referenceDecoded.data(), cacheWrapper->queryDecoded.data(), miOutput,
                uint32_t(cs), numLayersOutDecoder);
    } else {
        if (isMutualInformationData) {
            copyDecoderOutputSrnMutualInformation<<<sgl::uiceil(batchSize, 256), 256, 0, stream>>>(
                    cacheWrapper->queryDecoded.data(), miOutput, numLayersOutDecoder);
        } else if (calculateAbsoluteValue) {
            copyDecoderOutputSrnCorrelationCoefficientAbs<<<sgl::uiceil(batchSize, 256), 256, 0, stream>>>(
                    cacheWrapper->queryDecoded.data(), miOutput, numLayersOutDecoder);
        } else {
            copyDecoderOutputSrnCorrelationCoefficient<<<sgl::uiceil(batchSize, 256), 256, 0, stream>>>(
                    cacheWrapper->queryDecoded.data(), miOutput, numLayersOutDecoder);
        }
    }

    /*copySize = batchSize;
    float* data = new float[copySize];
    data[0] = 1000.0f;
    sgl::vk::checkCUresult(sgl::vk::g_cudaDeviceApiFunctionTable.hipMemcpyDtoHAsync(
            data, (hipDeviceptr_t)miOutput,
            sizeof(float) * copySize, stream), "Error in hipMemcpyDtoHAsync: ");
    sgl::vk::checkCUresult(sgl::vk::g_cudaDeviceApiFunctionTable.hipStreamSynchronize(
            stream), "Error in hipStreamSynchronize: ");
    std::cout << "miOutput:" << std::endl;
    for (int i = 0; i < testSize; i++) {
        std::cout << data[i];
        if (i != testSize - 1) {
            std::cout << ", ";
        }
        if (i % 20 == 19 && i != 0) {
            std::cout << std::endl;
        }
    }
    std::cout << std::endl;
    delete[] data;

    std::cout << std::endl << "END" << std::endl << std::endl;*/
}

void TinyCudaNNCorrelationCalculator::callbackBeginCompute() {
#ifdef TEST_INFERENCE_SPEED
    hipProfilerStart();
#endif
}

void TinyCudaNNCorrelationCalculator::callbackEndCompute() {
#ifdef TEST_INFERENCE_SPEED
    hipProfilerStop();
#endif
}
