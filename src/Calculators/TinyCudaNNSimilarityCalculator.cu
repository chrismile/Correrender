/*
 * BSD 2-Clause License
 *
 * Copyright (c) 2022, Christoph Neuhauser
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * * Redistributions of source code must retain the above copyright notice, this
 *   list of conditions and the following disclaimer.
 *
 * * Redistributions in binary form must reproduce the above copyright notice,
 *   this list of conditions and the following disclaimer in the documentation
 *   and/or other materials provided with the distribution.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <tiny-cuda-nn/trainer.h>
#include <tiny-cuda-nn/network_with_input_encoding.h>

#include <Math/Math.hpp>
#include <Utils/File/FileLoader.hpp>
#include <Utils/File/Archive.hpp>
#include <Utils/File/FileUtils.hpp>
#include <Graphics/Vulkan/Utils/InteropCuda.hpp>

#include "Volume/VolumeData.hpp"
#include "MutualInformation.cuh"
#include "TinyCudaNNSimilarityCalculator.hpp"

using precision_t = tcnn::network_precision_t;

struct TinyCudaNNModuleWrapper {
    nlohmann::json configEncoder;
    nlohmann::json configDecoder;
    std::shared_ptr<tcnn::Network<float, precision_t>> networkEncoder;
#ifdef TCNN_HALF_PRECISION
    std::shared_ptr<tcnn::Network<precision_t, precision_t>> networkEncoderHalf;
#endif
    std::shared_ptr<tcnn::Network<precision_t, precision_t>> networkDecoder;
};

struct TinyCudaNNCacheWrapper {
    tcnn::GPUMatrix<float> referenceInput;
#ifdef TCNN_HALF_PRECISION
    tcnn::GPUMatrix<precision_t> referenceInputHalf;
#endif
    tcnn::GPUMatrix<precision_t> referenceEncoded;
    tcnn::GPUMatrix<float> queryInput;
#ifdef TCNN_HALF_PRECISION
    tcnn::GPUMatrix<precision_t> queryInputHalf;
#endif
    tcnn::GPUMatrix<precision_t> queryEncoded;
    tcnn::GPUMatrix<precision_t> queryEncodedPermuted;
    tcnn::GPUMatrix<precision_t> symmetrizedReferenceInput;
    tcnn::GPUMatrix<precision_t> symmetrizedQueryInput;
    tcnn::GPUMatrix<precision_t> referenceDecoded;
    tcnn::GPUMatrix<precision_t> queryDecoded;
};

const uint32_t TINY_CUDA_NN_PARAMS_FORMAT_FLOAT = 0;
const uint32_t TINY_CUDA_NN_PARAMS_FORMAT_HALF = 1;
struct TinyCudaNNDataHeader {
    uint32_t format = 0;
    uint32_t numParams = 0;
};

TinyCudaNNSimilarityCalculator::TinyCudaNNSimilarityCalculator(sgl::vk::Renderer* renderer)
        : DeepLearningCudaSimilarityCalculator("tiny-cuda-nn", "tinyCudaNN", renderer) {
    cacheWrapper = std::make_shared<TinyCudaNNCacheWrapper>();
}

TinyCudaNNSimilarityCalculator::~TinyCudaNNSimilarityCalculator() {
}

template<class T, class PARAMS_T> static void loadNetwork(
        std::shared_ptr<tcnn::Network<T, PARAMS_T>>& network, const std::string& modelPath,
        const nlohmann::json& config, const sgl::ArchiveEntry& entry) {
    auto* header = reinterpret_cast<TinyCudaNNDataHeader*>(entry.bufferData.get());
    uint8_t* paramsData = entry.bufferData.get() + sizeof(TinyCudaNNDataHeader);
    uint32_t numParams = header->numParams;

    size_t sizePerEntry = header->format == TINY_CUDA_NN_PARAMS_FORMAT_FLOAT ? 4 : 2;
    if (numParams * sizePerEntry + sizeof(TinyCudaNNDataHeader) != entry.bufferSize) {
        sgl::Logfile::get()->throwError(
                "Error in loadNetwork: Invalid number of parameters for file size.");
    }

    bool hasInputEncoding = config.find("encoding") != config.end();
    bool isInputEncodingIdentity = false;
    auto encodingOpts = config.value("encoding", nlohmann::json::object());
    auto lossOpts = config.value("loss", nlohmann::json::object());
    auto optimizerOpts = config.value("optimizer", nlohmann::json::object());
    auto networkOpts = config.value("network", nlohmann::json::object());
    if (hasInputEncoding) {
        if (encodingOpts.value("otype", "Identity") == "Identity"
                && encodingOpts.value("scale", 1.0f) == 1.0f
                && encodingOpts.value("offset", 0.0f) == 0.0f) {
            isInputEncodingIdentity = true;
        }
    }

    uint32_t numInputDims = networkOpts["n_input_dims"];
    uint32_t numOutputDims = networkOpts["n_output_dims"];
    std::shared_ptr<tcnn::Loss<precision_t>> loss{tcnn::create_loss<precision_t>(lossOpts)};
    std::shared_ptr<tcnn::Optimizer<precision_t>> optimizer{tcnn::create_optimizer<precision_t>(optimizerOpts)};
    if (hasInputEncoding && !isInputEncodingIdentity) {
        std::shared_ptr<tcnn::NetworkWithInputEncoding<precision_t>> networkWithEnc =
                std::make_shared<tcnn::NetworkWithInputEncoding<precision_t>>(
                        numInputDims, numOutputDims, encodingOpts, networkOpts);
        if constexpr (std::is_same<T, float>::value) {
            network = std::static_pointer_cast<tcnn::Network<float, PARAMS_T>>(networkWithEnc);
        }
    } else {
        if constexpr (std::is_same<T, PARAMS_T>::value) {
            network = std::shared_ptr<tcnn::Network<PARAMS_T, PARAMS_T>>(
                    tcnn::create_network<PARAMS_T>(networkOpts));
        }
    }
    auto trainer = std::make_shared<tcnn::Trainer<T, PARAMS_T, PARAMS_T>>(network, optimizer, loss);

    // Do we need padding because the output width is not a multiple of 16?
    if (network->output_width() != network->padded_output_width()) {
        uint32_t numNeurons = networkOpts["n_neurons"];
        uint32_t paddingSize = numNeurons * (network->padded_output_width() - network->output_width());
        size_t numParamsOld = numParams;
        numParams += paddingSize;
        const uint8_t* paramsDataOld = paramsData;
        paramsData = new uint8_t[numParams * sizePerEntry];
        memcpy(paramsData, paramsDataOld, numParamsOld * sizePerEntry);
        memset(paramsData + numParamsOld * sizePerEntry, 0, paddingSize * sizePerEntry);
    }

    if (network->n_params() != numParams) {
        sgl::Logfile::get()->throwError(
                "Error in loadNetwork: Mismatching network parameter count (" + std::to_string(numParams)
                + " vs. " + std::to_string(network->n_params()) + ") for \"" + modelPath + "\".");
    }

#ifdef TCNN_HALF_PRECISION
    if (header->format == TINY_CUDA_NN_PARAMS_FORMAT_FLOAT) {
        trainer->set_params_full_precision(reinterpret_cast<float*>(paramsData), numParams, false);
    } else {
        trainer->set_params(reinterpret_cast<precision_t*>(paramsData), 0, false);
    }
#else
    if (header->format == TINY_CUDA_NN_PARAMS_FORMAT_FLOAT) {
        trainer->set_params(reinterpret_cast<float*>(paramsData), 0, false);
    } else {
        sgl::Logfile::get()->throwError(
                "Error in TinyCudaNNSimilarityCalculator::loadNetwork: Half precision build was disabled.");
    }
#endif

    if (network->output_width() != network->padded_output_width()) {
        delete[] paramsData;
    }

    // TODO: Support trainer->serialize()
}

void TinyCudaNNSimilarityCalculator::loadModelFromFile(const std::string& modelPath) {
    moduleWrapper = std::make_shared<TinyCudaNNModuleWrapper>();
    cacheWrapper = std::make_shared<TinyCudaNNCacheWrapper>();

    std::unordered_map<std::string, sgl::ArchiveEntry> archiveFiles;
    sgl::ArchiveFileLoadReturnType retVal = sgl::loadAllFilesFromArchive(modelPath, archiveFiles, true);
    if (retVal != sgl::ArchiveFileLoadReturnType::ARCHIVE_FILE_LOAD_SUCCESSFUL) {
        sgl::Logfile::get()->writeError(
                "Error in TinyCudaNNSimilarityCalculator::loadModelFromFile: Could not load data from model \""
                + modelPath + "\".");
        return;
    }

    auto itConfig = archiveFiles.find("config.json");
    auto itConfigEncoder = archiveFiles.find("config_encoder.json");
    auto itConfigDecoder = archiveFiles.find("config_decoder.json");
    if (itConfig != archiveFiles.end()) {
        const auto& entry = itConfig->second;
        moduleWrapper->configEncoder = moduleWrapper->configDecoder = nlohmann::json::parse(std::string(
                reinterpret_cast<char*>(entry.bufferData.get()), entry.bufferSize));
    } else if (itConfigEncoder != archiveFiles.end() && itConfigDecoder != archiveFiles.end()) {
        const auto& entryEncoder = itConfigEncoder->second;
        const auto& entryDecoder = itConfigDecoder->second;
        moduleWrapper->configEncoder = nlohmann::json::parse(std::string(
                reinterpret_cast<char*>(entryEncoder.bufferData.get()), entryEncoder.bufferSize));
        moduleWrapper->configDecoder = nlohmann::json::parse(std::string(
                reinterpret_cast<char*>(entryDecoder.bufferData.get()), entryDecoder.bufferSize));
    } else {
        sgl::Logfile::get()->writeError(
                "Error in TinyCudaNNSimilarityCalculator::loadModelFromFile: Could not load config from model \""
                + modelPath + "\".");
        return;
    }

    // Set input/output layer configurations for both networks.
    int es = volumeData->getEnsembleMemberCount();
    auto networkOpts = moduleWrapper->configEncoder.value("network", nlohmann::json::object());
    moduleWrapper->configEncoder["network"]["n_input_dims"] = 4;
    moduleWrapper->configDecoder["network"]["n_output_dims"] = 1;
    if (networkOpts.find("n_output_dims") == networkOpts.end()) {
        moduleWrapper->configEncoder["network"]["n_output_dims"] = moduleWrapper->configEncoder["network"]["n_neurons"];
        moduleWrapper->configDecoder["network"]["n_input_dims"] = moduleWrapper->configEncoder["network"]["n_neurons"];
    }

    bool hasInputEncoding = moduleWrapper->configEncoder.find("encoding") != moduleWrapper->configEncoder.end();
    bool isInputEncodingIdentity = false;
    auto encodingOpts = moduleWrapper->configEncoder.value("encoding", nlohmann::json::object());
    if (hasInputEncoding) {
        if (encodingOpts.value("otype", "Identity") == "Identity"
            && encodingOpts.value("scale", 1.0f) == 1.0f
            && encodingOpts.value("offset", 0.0f) == 0.0f) {
            isInputEncodingIdentity = true;
        }
    }

    auto itNetworkEncoder = archiveFiles.find("network_encoder.bin");
    auto itNetworkDecoder = archiveFiles.find("network_decoder.bin");
    if (itNetworkEncoder == archiveFiles.end()) {
        sgl::Logfile::get()->writeError(
                "Error in TinyCudaNNSimilarityCalculator::loadModelFromFile: Missing network_encoder.bin in file \""
                + modelPath + "\".");
        return;
    }
    if (itNetworkDecoder == archiveFiles.end()) {
        sgl::Logfile::get()->writeError(
                "Error in TinyCudaNNSimilarityCalculator::loadModelFromFile: Missing network_decoder.bin in file \""
                + modelPath + "\".");
        return;
    }
    moduleWrapper->networkEncoder = {};
#ifdef TCNN_HALF_PRECISION
    std::shared_ptr<tcnn::Network<precision_t, precision_t>> networkEncoderHalf;
#endif
    moduleWrapper->networkEncoderHalf = {};
    moduleWrapper->networkDecoder = {};
#ifdef TCNN_HALF_PRECISION
    if (hasInputEncoding && !isInputEncodingIdentity) {
#endif
        loadNetwork(moduleWrapper->networkEncoder, modelPath, moduleWrapper->configEncoder, itNetworkEncoder->second);
#ifdef TCNN_HALF_PRECISION
    } else {
        loadNetwork(moduleWrapper->networkEncoderHalf, modelPath, moduleWrapper->configEncoder, itNetworkEncoder->second);
    }
#endif
    loadNetwork(moduleWrapper->networkDecoder, modelPath, moduleWrapper->configDecoder, itNetworkDecoder->second);

    // numLayersInDecoder == numLayersOutEncoder when symmetrizer is sum operation.
    // TODO
#ifdef TCNN_HALF_PRECISION
    if (moduleWrapper->networkEncoderHalf) {
        numLayersInEncoder = uint32_t(moduleWrapper->networkEncoderHalf->input_width());
        numLayersOutEncoder = uint32_t(moduleWrapper->networkEncoderHalf->padded_output_width());
    } else {
        numLayersInEncoder = uint32_t(moduleWrapper->networkEncoder->input_width());
        numLayersOutEncoder = uint32_t(moduleWrapper->networkEncoder->padded_output_width());
    }
#endif
    numLayersInDecoder = uint32_t(moduleWrapper->networkDecoder->input_width());
    numLayersOutDecoder = uint32_t(moduleWrapper->networkDecoder->padded_output_width());
    //moduleWrapper->networkDecoder->channelsIn():
}

void TinyCudaNNSimilarityCalculator::recreateCache(int batchSize) {
    int es = volumeData->getEnsembleMemberCount();

    //cacheWrapper->referenceInput = tcnn::GPUMatrix<float>(uint32_t(es) * 4, 1);
    //cacheWrapper->referenceEncoded = tcnn::GPUMatrix<precision_t>(numLayersOutEncoder, 1);
    //cacheWrapper->queryInput = tcnn::GPUMatrix<float>(uint32_t(es) * 4, batchSize);
    //cacheWrapper->queryEncoded = tcnn::GPUMatrix<precision_t>(numLayersOutEncoder, batchSize);
    //cacheWrapper->queryEncodedPermuted = tcnn::GPUMatrix<precision_t>(numLayersOutEncoder, batchSize);
    //cacheWrapper->symmetrizedReferenceInput = tcnn::GPUMatrix<precision_t>(numLayersInDecoder, batchSize);
    //cacheWrapper->symmetrizedQueryInput = tcnn::GPUMatrix<precision_t>(numLayersInDecoder, batchSize);
    //cacheWrapper->referenceDecoded = tcnn::GPUMatrix<precision_t>(numLayersOutDecoder, batchSize);
    //cacheWrapper->queryDecoded = tcnn::GPUMatrix<precision_t>(numLayersOutDecoder, batchSize);

    cacheWrapper->referenceInput = tcnn::GPUMatrix<float>();
#ifdef TCNN_HALF_PRECISION
    cacheWrapper->referenceInputHalf = tcnn::GPUMatrix<precision_t>();
#endif
    cacheWrapper->referenceEncoded = tcnn::GPUMatrix<precision_t>();
    cacheWrapper->queryInput = tcnn::GPUMatrix<float>();
#ifdef TCNN_HALF_PRECISION
    cacheWrapper->queryInputHalf = tcnn::GPUMatrix<precision_t>();
#endif
    cacheWrapper->queryEncoded = tcnn::GPUMatrix<precision_t>();
    cacheWrapper->queryEncodedPermuted = tcnn::GPUMatrix<precision_t>();
    cacheWrapper->symmetrizedReferenceInput = tcnn::GPUMatrix<precision_t>();
    cacheWrapper->symmetrizedQueryInput = tcnn::GPUMatrix<precision_t>();
    cacheWrapper->referenceDecoded = tcnn::GPUMatrix<precision_t>();
    cacheWrapper->queryDecoded = tcnn::GPUMatrix<precision_t>();

    // mlp_fused_forward needs multiple of 16 for number of input layers.
    uint32_t numInputLayers = 16;
    uint32_t referenceInputBatchSize =
            sgl::uiceil(uint32_t(es), tcnn::batch_size_granularity) * tcnn::batch_size_granularity;
#ifdef TCNN_HALF_PRECISION
    if (moduleWrapper->networkEncoderHalf) {
        cacheWrapper->referenceInputHalf = tcnn::GPUMatrix<precision_t>(numInputLayers, referenceInputBatchSize);
        cacheWrapper->queryInputHalf = tcnn::GPUMatrix<precision_t>(numInputLayers, uint32_t(es) * batchSize);
    }
#endif
    cacheWrapper->referenceInput = tcnn::GPUMatrix<float>(numInputLayers, referenceInputBatchSize);
    cacheWrapper->queryInput = tcnn::GPUMatrix<float>(numInputLayers, uint32_t(es) * batchSize);
    cacheWrapper->referenceEncoded = tcnn::GPUMatrix<precision_t>(numLayersOutEncoder, referenceInputBatchSize);
    cacheWrapper->queryEncoded = tcnn::GPUMatrix<precision_t>(numLayersOutEncoder, uint32_t(es) * batchSize);
    cacheWrapper->queryEncodedPermuted = tcnn::GPUMatrix<precision_t>(numLayersOutEncoder, uint32_t(es) * batchSize);
    cacheWrapper->symmetrizedReferenceInput = tcnn::GPUMatrix<precision_t>(numLayersInDecoder, uint32_t(es) * batchSize);
    cacheWrapper->symmetrizedQueryInput = tcnn::GPUMatrix<precision_t>(numLayersInDecoder, uint32_t(es) * batchSize);
    cacheWrapper->referenceDecoded = tcnn::GPUMatrix<precision_t>(numLayersOutDecoder, uint32_t(es) * batchSize);
    cacheWrapper->queryDecoded = tcnn::GPUMatrix<precision_t>(numLayersOutDecoder, uint32_t(es) * batchSize);
}

hipDeviceptr_t TinyCudaNNSimilarityCalculator::getReferenceInputPointer()  {
    return reinterpret_cast<hipDeviceptr_t>(cacheWrapper->referenceInput.data());
}

hipDeviceptr_t TinyCudaNNSimilarityCalculator::getQueryInputPointer()  {
    return reinterpret_cast<hipDeviceptr_t>(cacheWrapper->queryInput.data());
}

void TinyCudaNNSimilarityCalculator::runInferenceReference() {
#ifdef TCNN_HALF_PRECISION
    if (moduleWrapper->networkEncoderHalf) {
        uint32_t arraySize = cacheWrapper->referenceInputHalf.n() * cacheWrapper->referenceInputHalf.m();
        convertFloatToHalfArray<<<sgl::uiceil(arraySize, 256), 256, 0, stream>>>(
                cacheWrapper->referenceInputHalf.data(), cacheWrapper->referenceInput.data(), arraySize);
        moduleWrapper->networkEncoderHalf->inference_mixed_precision(
                stream, cacheWrapper->referenceInputHalf, cacheWrapper->referenceEncoded);
    } else {
        moduleWrapper->networkEncoder->inference_mixed_precision(
                stream, cacheWrapper->referenceInput, cacheWrapper->referenceEncoded);
    }
#else
    moduleWrapper->networkEncoder->inference(
                stream, cacheWrapper->referenceInput, cacheWrapper->referenceEncoded);
#endif
}

void TinyCudaNNSimilarityCalculator::runInferenceBatch(uint32_t batchOffset, uint32_t batchSize)  {
    int es = volumeData->getEnsembleMemberCount();

#ifdef TCNN_HALF_PRECISION
    if (moduleWrapper->networkEncoderHalf) {
        uint32_t arraySize = cacheWrapper->queryInputHalf.n() * cacheWrapper->queryInputHalf.m();
        convertFloatToHalfArray<<<sgl::uiceil(batchSize, 256), 256, 0, stream>>>(
                cacheWrapper->queryInputHalf.data(), cacheWrapper->queryInput.data(), arraySize);
        moduleWrapper->networkEncoderHalf->inference_mixed_precision(
                stream, cacheWrapper->queryInputHalf, cacheWrapper->queryEncoded);
    } else {
        moduleWrapper->networkEncoder->inference_mixed_precision(
                stream, cacheWrapper->queryInput, cacheWrapper->queryEncoded);
    }
#else
    moduleWrapper->networkEncoder->inference(
                stream, cacheWrapper->queryInput, cacheWrapper->queryEncoded);
#endif

    sgl::vk::checkCUresult(sgl::vk::g_cudaDeviceApiFunctionTable.cuMemcpyAsync(
            (hipDeviceptr_t)cacheWrapper->queryEncodedPermuted.data(),
            (hipDeviceptr_t)cacheWrapper->queryEncoded.data(),
            sizeof(float) * batchSize, stream), "Error in cuMemcpyAsync: ");
    randomShuffleFisherYatesXorshift<<<sgl::uiceil(batchSize, 256), 256, 0, stream>>>(
            cacheWrapper->queryEncodedPermuted.data(), numLayersOutEncoder);

    symmetrizer<<<sgl::uiceil(batchSize, 256), 256, 0, stream>>>(
            cacheWrapper->referenceEncoded.data(), cacheWrapper->queryEncoded.data(),
            cacheWrapper->symmetrizedReferenceInput.data(), numLayersOutEncoder);
    symmetrizer<<<sgl::uiceil(batchSize, 256), 256, 0, stream>>>(
            cacheWrapper->referenceEncoded.data(), cacheWrapper->queryEncodedPermuted.data(),
            cacheWrapper->symmetrizedQueryInput.data(), numLayersOutEncoder);

#ifdef TCNN_HALF_PRECISION
    moduleWrapper->networkDecoder->inference_mixed_precision(
            stream, cacheWrapper->symmetrizedReferenceInput, cacheWrapper->referenceDecoded);
    moduleWrapper->networkDecoder->inference_mixed_precision(
            stream, cacheWrapper->symmetrizedQueryInput, cacheWrapper->queryDecoded);
#else
    moduleWrapper->networkDecoder->inference(
            stream, cacheWrapper->symmetrizedReferenceInput, cacheWrapper->referenceDecoded);
    moduleWrapper->networkDecoder->inference(
            stream, cacheWrapper->symmetrizedQueryInput, cacheWrapper->queryDecoded);
#endif

    float *miOutput = reinterpret_cast<float*>(outputImageBufferCu) + batchOffset;
    combineDecoderOutput<<<sgl::uiceil(batchSize, 256), 256, 0, stream>>>(
            cacheWrapper->referenceDecoded.data(), cacheWrapper->queryDecoded.data(), miOutput,
            uint32_t(es), numLayersOutDecoder);
}
